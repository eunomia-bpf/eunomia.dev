#include "hip/hip_runtime.h"
// Copyright 2021-2022 NVIDIA Corporation. All rights reserved
//
// The sample provides workflow for adding nested ranges for profiling with CUPTI profiling APIs.
// The psuedo code for the sample
// cuptiProfilerPushRange(rangeA)           // push rangeA -> nesting level 1
//    launch kernel A
//    cuptiProfilerPushRange(rangeB)        // push rangeB -> nesting level 2
//        launch kernel B
//    cuptiProfilerPopRange()               // pop rangeB
// cuptiProfilerPopRange()                  // pop rangeA
//
// Notes:
// 1) Even though nested ranges are subset of user range they still count as individual range,
//    so the number of ranges need to be sum of user ranges and nested ranges and as the ranges number increases the profiling time also increases.
// 2) The number of passes required for collecting counter data will vary with number of nesting level used.
//    e.g. "sm__mio_inst_issued.sum" metric in GA100 needs 2 passes for collecting the counter data with no nesting (nestingLevel = 1).
//    if we add a nested range (nestingLevel = 2) then the number of passes required will be 2 times. (4 passes)
//    (You can refer to cupti_metric_properties sample for getting the metrics properties like number of passes required and
//    which type of metric it is (HW/SW) for a particular chip)
//

// Standard STL headers
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <cstring>

// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// CUPTI headers
#include "helper_cupti.h"
#include <cupti_target.h>
#include <cupti_profiler_target.h>

// NVPW headers
#include <nvperf_host.h>

// Make use of example code wrappers for NVPW calls
#include <Eval.h>
#include <Utils.h>
#include <Metric.h>
#include <FileOp.h>

// Global emums and variables
static int s_NumRanges = 2;
static int s_NumNestingLevels = 2;

enum class eVectorOperationType
{
    VEC_ADD,
    VEC_SUB
};

// Macros
#define DEFAULT_METRIC_NAME "sm__ctas_launched.sum"

// Kernels
__global__ void
VectorAdd(
    const int *pA,
    const int *pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
    {
        pC[i] = pA[i] + pB[i];
    }
}

__global__ void
VectorSubtract(
    const int *pA,
    const int *pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
    {
        pC[i] = pA[i] - pB[i];
    }
}


static void
InitializeVector(
    int *pVector,
    int N)
{
    for (int i = 0; i < N; i++)
    {
        pVector[i] = i;
    }
}

static void
CleanUp(
    int *pHostA,
    int *pHostB,
    int *pHostC,
    int *pDeviceA,
    int *pDeviceB,
    int *pDeviceC)
{
    // Free host memory.
    if (pHostA)
    {
        free(pHostA);
    }
    if (pHostB)
    {
        free(pHostB);
    }
    if (pHostC)
    {
        free(pHostC);
    }

    // Free device memory.
    if (pDeviceA)
    {
        RUNTIME_API_CALL(hipFree(pDeviceA));
    }
    if (pDeviceB)
    {
        RUNTIME_API_CALL(hipFree(pDeviceB));
    }
    if (pDeviceC)
    {
        RUNTIME_API_CALL(hipFree(pDeviceC));
    }
}

static void
ProcessVector(
    int numOfElements,
    eVectorOperationType operationType)
{
    size_t size = numOfElements * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int *pHostA, *pHostB, *pHostC;
    int *pDeviceA, *pDeviceB, *pDeviceC;
    int i, res;

    // Allocate input vectors pHostA and pHostB in host memory.
    pHostA = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostA);

    pHostB = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostB);

    pHostC = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostC);

    // Initialize input vectors.
    InitializeVector(pHostA, numOfElements);
    InitializeVector(pHostB, numOfElements);
    memset(pHostC, 0, size);

    // Allocate vectors in device memory.
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceA, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceB, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceC, size));

    // Copy vectors from host memory to device memory.
    RUNTIME_API_CALL(hipMemcpy(pDeviceA, pHostA, size, hipMemcpyHostToDevice));
    RUNTIME_API_CALL(hipMemcpy(pDeviceB, pHostB, size, hipMemcpyHostToDevice));

    // Invoke kernel.
    threadsPerBlock = 256;
    blocksPerGrid = (numOfElements + threadsPerBlock - 1) / threadsPerBlock;

    if (operationType == eVectorOperationType::VEC_ADD)
    {
        printf("Launching VecAdd kernel: blocks %d, thread/block %d\n", blocksPerGrid, threadsPerBlock);
        VectorAdd <<< blocksPerGrid, threadsPerBlock >>> (pDeviceA, pDeviceB, pDeviceC, numOfElements);
        RUNTIME_API_CALL(hipGetLastError());
    }

    if (operationType == eVectorOperationType::VEC_SUB)
    {
        printf("Launching VecSub kernel: blocks %d, thread/block %d\n", blocksPerGrid, threadsPerBlock);
        VectorSubtract <<< blocksPerGrid, threadsPerBlock >>> (pDeviceA, pDeviceB, pDeviceC, numOfElements);
        RUNTIME_API_CALL(hipGetLastError());
    }

    // Copy result from device memory to host memory.
    // pHostC contains the result in host memory.
    RUNTIME_API_CALL(hipMemcpy(pHostC, pDeviceC, size, hipMemcpyDeviceToHost));

    // Verify result
    for (i = 0; i < numOfElements; ++i)
    {
        if (operationType == eVectorOperationType::VEC_ADD)
            res = pHostA[i] + pHostB[i];

        if (operationType == eVectorOperationType::VEC_SUB)
            res = pHostA[i] - pHostB[i];

        if (pHostC[i] != res)
        {
            fprintf(stderr, "error: result verification failed\n");
            exit(EXIT_FAILURE);
        }
    }

    CleanUp(pHostA, pHostB, pHostC, pDeviceA, pDeviceB, pDeviceC);
}

bool CreateCounterDataImage(
    std::vector<uint8_t>& counterDataImage,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImagePrefix)
{
    CUpti_Profiler_CounterDataImageOptions counterDataImageOptions;
    counterDataImageOptions.pCounterDataPrefix = &counterDataImagePrefix[0];
    counterDataImageOptions.counterDataPrefixSize = counterDataImagePrefix.size();
    counterDataImageOptions.maxNumRanges = s_NumRanges;
    counterDataImageOptions.maxNumRangeTreeNodes = s_NumRanges;
    counterDataImageOptions.maxRangeNameLength = 64;

    CUpti_Profiler_CounterDataImage_CalculateSize_Params calculateSizeParams = {CUpti_Profiler_CounterDataImage_CalculateSize_Params_STRUCT_SIZE};
    calculateSizeParams.pOptions = &counterDataImageOptions;
    calculateSizeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateSize(&calculateSizeParams));

    CUpti_Profiler_CounterDataImage_Initialize_Params initializeParams = {CUpti_Profiler_CounterDataImage_Initialize_Params_STRUCT_SIZE};
    initializeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    initializeParams.pOptions = &counterDataImageOptions;
    initializeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    counterDataImage.resize(calculateSizeParams.counterDataImageSize);
    initializeParams.pCounterDataImage = &counterDataImage[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitialize(&initializeParams));

    CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params scratchBufferSizeParams = {CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params_STRUCT_SIZE};
    scratchBufferSizeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    scratchBufferSizeParams.pCounterDataImage = initializeParams.pCounterDataImage;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateScratchBufferSize(&scratchBufferSizeParams));

    counterDataScratchBuffer.resize(scratchBufferSizeParams.counterDataScratchBufferSize);
    CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params initScratchBufferParams = {CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params_STRUCT_SIZE};
    initScratchBufferParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    initScratchBufferParams.pCounterDataImage = initializeParams.pCounterDataImage;
    initScratchBufferParams.counterDataScratchBufferSize = scratchBufferSizeParams.counterDataScratchBufferSize;
    initScratchBufferParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitializeScratchBuffer(&initScratchBufferParams));

    return true;
}

bool RunTest(
    std::vector<uint8_t>& configImage,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImage,
    CUpti_ProfilerReplayMode profilerReplayMode,
    CUpti_ProfilerRange profilerRange)
{
    hipCtx_t cuContext;
    DRIVER_API_CALL(hipCtxGetCurrent(&cuContext));

    CUpti_Profiler_BeginSession_Params beginSessionParams = { CUpti_Profiler_BeginSession_Params_STRUCT_SIZE };
    beginSessionParams.ctx = cuContext;
    beginSessionParams.counterDataImageSize = counterDataImage.size();
    beginSessionParams.pCounterDataImage = &counterDataImage[0];
    beginSessionParams.counterDataScratchBufferSize = counterDataScratchBuffer.size();
    beginSessionParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];
    beginSessionParams.range = profilerRange;
    beginSessionParams.replayMode = profilerReplayMode;
    beginSessionParams.maxRangesPerPass = s_NumRanges;
    beginSessionParams.maxLaunchesPerPass = s_NumRanges;
    CUPTI_API_CALL(cuptiProfilerBeginSession(&beginSessionParams));

    CUpti_Profiler_SetConfig_Params setConfigParams = { CUpti_Profiler_SetConfig_Params_STRUCT_SIZE };
    setConfigParams.pConfig = &configImage[0];
    setConfigParams.configSize = configImage.size();
    setConfigParams.passIndex = 0;
    setConfigParams.minNestingLevel = 1;
    setConfigParams.numNestingLevels = s_NumNestingLevels;
    CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));

    // User takes the resposiblity of replaying the kernel launches.
    CUpti_Profiler_BeginPass_Params beginPassParams = { CUpti_Profiler_BeginPass_Params_STRUCT_SIZE };
    CUpti_Profiler_EndPass_Params endPassParams = { CUpti_Profiler_EndPass_Params_STRUCT_SIZE };
    do
    {
        CUPTI_API_CALL(cuptiProfilerBeginPass(&beginPassParams));

        {
            CUpti_Profiler_EnableProfiling_Params enableProfilingParams = { CUpti_Profiler_EnableProfiling_Params_STRUCT_SIZE };
            CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));

            CUpti_Profiler_PushRange_Params pushRangeParams = { CUpti_Profiler_PushRange_Params_STRUCT_SIZE };
            pushRangeParams.pRangeName = "userRangeA";
            printf("\nStart of userRangeA\n");
            CUPTI_API_CALL(cuptiProfilerPushRange(&pushRangeParams));
            {
                ProcessVector(50000, eVectorOperationType::VEC_ADD);
                // Nested range start.
                pushRangeParams.pRangeName = "userRangeB";
                printf("Start of userRangeB\n");
                CUPTI_API_CALL(cuptiProfilerPushRange(&pushRangeParams));
                {
                    ProcessVector(10000, eVectorOperationType::VEC_SUB);
                }
                CUpti_Profiler_PopRange_Params popRangeParams = { CUpti_Profiler_PopRange_Params_STRUCT_SIZE };
                printf("End of userRangeB\n");
                CUPTI_API_CALL(cuptiProfilerPopRange(&popRangeParams));
                // Nested range End.
            }

            CUpti_Profiler_PopRange_Params popRangeParams = { CUpti_Profiler_PopRange_Params_STRUCT_SIZE };
            printf("End of userRangeA\n");
            CUPTI_API_CALL(cuptiProfilerPopRange(&popRangeParams));

            CUpti_Profiler_DisableProfiling_Params disableProfilingParams = { CUpti_Profiler_DisableProfiling_Params_STRUCT_SIZE };
            CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));
        }

        CUPTI_API_CALL(cuptiProfilerEndPass(&endPassParams));

    } while (!endPassParams.allPassesSubmitted);

    CUpti_Profiler_FlushCounterData_Params flushCounterDataParams = {CUpti_Profiler_FlushCounterData_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerFlushCounterData(&flushCounterDataParams));

    CUpti_Profiler_UnsetConfig_Params unsetConfigParams = {CUpti_Profiler_UnsetConfig_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerUnsetConfig(&unsetConfigParams));

    CUpti_Profiler_EndSession_Params endSessionParams = {CUpti_Profiler_EndSession_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerEndSession(&endSessionParams));

    return true;
}

int main(
    int argc,
    char *argv[])
{
    hipDevice_t cuDevice;
    std::vector<std::string> metricNames;
    std::vector<uint8_t> counterDataImagePrefix;
    std::vector<uint8_t> configImage;
    std::vector<uint8_t> counterDataImage;
    std::vector<uint8_t> counterDataScratchBuffer;
    std::vector<uint8_t> counterAvailabilityImage;
    std::string CounterDataFileName("SimpleCupti.counterdata");
    std::string CounterDataSBFileName("SimpleCupti.counterdataSB");
    CUpti_ProfilerReplayMode profilerReplayMode = CUPTI_UserReplay;
    CUpti_ProfilerRange profilerRange = CUPTI_UserRange;
    char* metricName;
    int deviceCount, deviceNum;
    int computeCapabilityMajor = 0, computeCapabilityMinor = 0;

    printf("Usage: %s [device_num] [metric_names comma separated]\n", argv[0]);

    DRIVER_API_CALL(hipInit(0));
    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        printf("There is no device supporting CUDA.\n");
        exit(EXIT_WAIVED);
    }

    if (argc > 1)
    {
        deviceNum = atoi(argv[1]);
    }
    else
    {
        deviceNum = 0;
    }
    printf("CUDA Device Number: %d\n", deviceNum);

    DRIVER_API_CALL(hipDeviceGet(&cuDevice, deviceNum));
    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));

    printf("Compute Capability of Device: %d.%d\n", computeCapabilityMajor,computeCapabilityMinor);

    // Initialize profiler API and test device compatibility.
    CUpti_Profiler_Initialize_Params profilerInitializeParams = { CUpti_Profiler_Initialize_Params_STRUCT_SIZE };
    CUPTI_API_CALL(cuptiProfilerInitialize(&profilerInitializeParams));
    CUpti_Profiler_DeviceSupported_Params params = { CUpti_Profiler_DeviceSupported_Params_STRUCT_SIZE };
    params.cuDevice = deviceNum;
    params.api = CUPTI_PROFILER_RANGE_PROFILING;
    CUPTI_API_CALL(cuptiProfilerDeviceSupported(&params));

    if (params.isSupported != CUPTI_PROFILER_CONFIGURATION_SUPPORTED)
    {
        ::std::cerr << "Unable to profile on device " << deviceNum << ::std::endl;

        if (params.architecture == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice architecture is not supported" << ::std::endl;
        }

        if (params.sli == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice sli configuration is not supported" << ::std::endl;
        }

        if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice vgpu configuration is not supported" << ::std::endl;
        }
        else if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_DISABLED)
        {
            ::std::cerr << "\tdevice vgpu configuration disabled profiling support" << ::std::endl;
        }

        if (params.confidentialCompute == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice confidential compute configuration is not supported" << ::std::endl;
        }

        if (params.cmp == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tNVIDIA Crypto Mining Processors (CMP) are not supported" << ::std::endl;
        }

        if (params.wsl == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tWSL is not supported" << ::std::endl;
        }
        exit(EXIT_WAIVED);
    }

    // Get the names of the metrics to collect.
    if (argc > 2)
    {
        metricName = strtok(argv[2], ",");
        while(metricName != NULL)
        {
            metricNames.push_back(metricName);
            metricName = strtok(NULL, ",");
        }
    }
    else {
        metricNames.push_back(DEFAULT_METRIC_NAME);
    }

    hipCtx_t cuContext;
    DRIVER_API_CALL(hipCtxCreate(&cuContext, 0, cuDevice));

    // Get chip name for the cuda  device.
    CUpti_Device_GetChipName_Params getChipNameParams = { CUpti_Device_GetChipName_Params_STRUCT_SIZE };
    getChipNameParams.deviceIndex = deviceNum;
    CUPTI_API_CALL(cuptiDeviceGetChipName(&getChipNameParams));
    std::string chipName(getChipNameParams.pChipName);

    CUpti_Profiler_GetCounterAvailability_Params getCounterAvailabilityParams = { CUpti_Profiler_GetCounterAvailability_Params_STRUCT_SIZE };
    getCounterAvailabilityParams.ctx = cuContext;
    CUPTI_API_CALL(cuptiProfilerGetCounterAvailability(&getCounterAvailabilityParams));

    counterAvailabilityImage.clear();
    counterAvailabilityImage.resize(getCounterAvailabilityParams.counterAvailabilityImageSize);
    getCounterAvailabilityParams.pCounterAvailabilityImage = counterAvailabilityImage.data();
    CUPTI_API_CALL(cuptiProfilerGetCounterAvailability(&getCounterAvailabilityParams));

    // Generate configuration for metrics, this can also be done offline.
    NVPW_InitializeHost_Params initializeHostParams = { NVPW_InitializeHost_Params_STRUCT_SIZE };
    RETURN_IF_NVPW_ERROR(0, NVPW_InitializeHost(&initializeHostParams));

    if (metricNames.size())
    {
        if (!NV::Metric::Config::GetConfigImage(chipName, metricNames, configImage, counterAvailabilityImage.data()))
        {
            std::cerr << "Failed to create the ConfigImage." << std::endl;
            exit(EXIT_FAILURE);
        }
        if (!NV::Metric::Config::GetCounterDataPrefixImage(chipName, metricNames, counterDataImagePrefix))
        {
            std::cerr << "Failed to create the CounterDataPrefixImage." << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    else
    {
        std::cerr << "No metrics provided to profile." << std::endl;
        exit(EXIT_FAILURE);
    }

    if (!CreateCounterDataImage(counterDataImage, counterDataScratchBuffer, counterDataImagePrefix))
    {
        std::cerr << "  Failed to create the CounterDataImage." << std::endl;
        exit(EXIT_FAILURE);
    }

    if (!RunTest(configImage, counterDataScratchBuffer, counterDataImage, profilerReplayMode, profilerRange))
    {
        std::cerr << "Failed to run the sample." << std::endl;
        exit(EXIT_FAILURE);
    }

    CUpti_Profiler_DeInitialize_Params profilerDeInitializeParams = {CUpti_Profiler_DeInitialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerDeInitialize(&profilerDeInitializeParams));
    DRIVER_API_CALL(hipCtxDestroy(cuContext));

    // Dump counterDataImage in file.
    WriteBinaryFile(CounterDataFileName.c_str(), counterDataImage);
    WriteBinaryFile(CounterDataSBFileName.c_str(), counterDataScratchBuffer);

    // Evaluation of metrics collected in counterDataImage, this can also be done offline.
    NV::Metric::Eval::PrintMetricValues(chipName, counterDataImage, metricNames);
    exit(EXIT_SUCCESS);
}
