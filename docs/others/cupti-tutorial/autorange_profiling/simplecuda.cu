#include "hip/hip_runtime.h"
#include <cupti_target.h>
#include <cupti_profiler_target.h>
#include <nvperf_host.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <Metric.h>
#include <Eval.h>
#include <FileOp.h>

#define NVPW_API_CALL(apiFuncCall)                                             \
do {                                                                           \
    NVPA_Status _status = apiFuncCall;                                         \
    if (_status != NVPA_STATUS_SUCCESS) {                                      \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define CUPTI_API_CALL(apiFuncCall)                                            \
do {                                                                           \
    CUptiResult _status = apiFuncCall;                                         \
    if (_status != CUPTI_SUCCESS) {                                            \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

static int numRanges = 2;
#define METRIC_NAME "smsp__warps_launched.avg+"

// Device code
  __global__ void VecAdd(const int* A, const int* B, int* C, int N)
 {
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     if (i < N)
         C[i] = A[i] + B[i];
 }

 // Device code
  __global__ void VecSub(const int* A, const int* B, int* C, int N)
 {
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     if (i < N)
         C[i] = A[i] - B[i];
 }


static void initVec(int *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}

static void cleanUp(int *h_A, int *h_B, int *h_C, int *h_D, int *d_A, int *d_B, int *d_C, int *d_D)
{
  if (d_A)
    hipFree(d_A);
  if (d_B)
    hipFree(d_B);
  if (d_C)
    hipFree(d_C);
  if (d_D)
    hipFree(d_D);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_B)
    free(h_B);
  if (h_C)
    free(h_C);
  if (h_D)
    free(h_D);
}

static void VectorAddSubtract()
{
  int N = 50000;
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int *h_A, *h_B, *h_C, *h_D;
  int *d_A, *d_B, *d_C, *d_D;
  int i, sum, diff;

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);
  h_D = (int*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);
  memset(h_D, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);
  hipMalloc((void**)&d_D, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  printf("Launching kernel: blocks %d, thread/block %d\n",
         blocksPerGrid, threadsPerBlock);

  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  VecSub<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_D, N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);

  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    diff = h_A[i] - h_B[i];
    if (h_C[i] != sum || h_D[i] != diff) {
      fprintf(stderr, "error: result verification failed\n");
      exit(-1);
    }
  }

  cleanUp(h_A, h_B, h_C, h_D, d_A, d_B, d_C, d_D);
}

bool CreateCounterDataImage(
    std::vector<uint8_t>& counterDataImage,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImagePrefix)
{

    CUpti_Profiler_CounterDataImageOptions counterDataImageOptions;
    counterDataImageOptions.pCounterDataPrefix = &counterDataImagePrefix[0];
    counterDataImageOptions.counterDataPrefixSize = counterDataImagePrefix.size();
    counterDataImageOptions.maxNumRanges = numRanges;
    counterDataImageOptions.maxNumRangeTreeNodes = numRanges;
    counterDataImageOptions.maxRangeNameLength = 64;

    CUpti_Profiler_CounterDataImage_CalculateSize_Params calculateSizeParams = {CUpti_Profiler_CounterDataImage_CalculateSize_Params_STRUCT_SIZE};

    calculateSizeParams.pOptions = &counterDataImageOptions;
    calculateSizeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;

    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateSize(&calculateSizeParams));

    CUpti_Profiler_CounterDataImage_Initialize_Params initializeParams = {CUpti_Profiler_CounterDataImage_Initialize_Params_STRUCT_SIZE};
    initializeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    initializeParams.pOptions = &counterDataImageOptions;
    initializeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;

    counterDataImage.resize(calculateSizeParams.counterDataImageSize);
    initializeParams.pCounterDataImage = &counterDataImage[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitialize(&initializeParams));

    CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params scratchBufferSizeParams = {CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params_STRUCT_SIZE};
    scratchBufferSizeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    scratchBufferSizeParams.pCounterDataImage = initializeParams.pCounterDataImage;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateScratchBufferSize(&scratchBufferSizeParams));

    counterDataScratchBuffer.resize(scratchBufferSizeParams.counterDataScratchBufferSize);

    CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params initScratchBufferParams = {CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params_STRUCT_SIZE};
    initScratchBufferParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;

    initScratchBufferParams.pCounterDataImage = initializeParams.pCounterDataImage;
    initScratchBufferParams.counterDataScratchBufferSize = scratchBufferSizeParams.counterDataScratchBufferSize;
    initScratchBufferParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];

    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitializeScratchBuffer(&initScratchBufferParams));

    return true;
}

bool runTest(int deviceNum,
                       std::vector<uint8_t>& configImage,
                       std::vector<uint8_t>& counterDataScratchBuffer,
                       std::vector<uint8_t>& counterDataImage,
                       CUpti_ProfilerReplayMode profilerReplayMode,
                       CUpti_ProfilerRange profilerRange)
{

    hipDevice_t cuDevice;
    DRIVER_API_CALL(hipDeviceGet(&cuDevice, deviceNum));

    hipCtx_t cuContext;
    DRIVER_API_CALL(hipCtxCreate(&cuContext, 0, cuDevice));

    CUpti_Profiler_BeginSession_Params beginSessionParams = {CUpti_Profiler_BeginSession_Params_STRUCT_SIZE};
    CUpti_Profiler_SetConfig_Params setConfigParams = {CUpti_Profiler_SetConfig_Params_STRUCT_SIZE};
    CUpti_Profiler_EnableProfiling_Params enableProfilingParams = {CUpti_Profiler_EnableProfiling_Params_STRUCT_SIZE};
    CUpti_Profiler_DisableProfiling_Params disableProfilingParams = {CUpti_Profiler_DisableProfiling_Params_STRUCT_SIZE};

    beginSessionParams.ctx = NULL;
    beginSessionParams.counterDataImageSize = counterDataImage.size();
    beginSessionParams.pCounterDataImage = &counterDataImage[0];
    beginSessionParams.counterDataScratchBufferSize = counterDataScratchBuffer.size();
    beginSessionParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];
    beginSessionParams.range = profilerRange;
    beginSessionParams.replayMode = profilerReplayMode;
    beginSessionParams.maxRangesPerPass = numRanges;
    beginSessionParams.maxLaunchesPerPass = numRanges;

    CUPTI_API_CALL(cuptiProfilerBeginSession(&beginSessionParams));

    setConfigParams.pConfig = &configImage[0];
    setConfigParams.configSize = configImage.size();

    if(profilerReplayMode == CUPTI_KernelReplay)    /* Profile in KernelReplayMode */
    {
        setConfigParams.passIndex = 0;
        CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));
        CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
        VectorAddSubtract();
        CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));
    }
    else if(profilerReplayMode == CUPTI_UserReplay) /* Profiler in UserReplayMode */
    {
        setConfigParams.passIndex = 0;
        CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));
        /* User takes the resposiblity of replaying the kernel launches */
        CUpti_Profiler_BeginPass_Params beginPassParams = {CUpti_Profiler_BeginPass_Params_STRUCT_SIZE};
        CUpti_Profiler_EndPass_Params endPassParams = {CUpti_Profiler_EndPass_Params_STRUCT_SIZE};
        do
        {
            CUPTI_API_CALL(cuptiProfilerBeginPass(&beginPassParams));
            {
                CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
                VectorAddSubtract();
                CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));
            }
            CUPTI_API_CALL(cuptiProfilerEndPass(&endPassParams));
        }while(!endPassParams.allPassesSubmitted);
        CUpti_Profiler_FlushCounterData_Params flushCounterDataParams = {CUpti_Profiler_FlushCounterData_Params_STRUCT_SIZE};
        CUPTI_API_CALL(cuptiProfilerFlushCounterData(&flushCounterDataParams));
    }
    CUpti_Profiler_UnsetConfig_Params unsetConfigParams = {CUpti_Profiler_UnsetConfig_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerUnsetConfig(&unsetConfigParams));
    CUpti_Profiler_EndSession_Params endSessionParams = {CUpti_Profiler_EndSession_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerEndSession(&endSessionParams));

    DRIVER_API_CALL(hipCtxDestroy(cuContext));

    return true;
}

int main(int argc, char* argv[])
{
    std::vector<std::string> metricNames;
    std::vector<uint8_t> counterDataImagePrefix;
    std::vector<uint8_t> configImage;
    std::vector<uint8_t> counterDataImage;
    std::vector<uint8_t> counterDataScratchBuffer;
    std::string CounterDataFileName("SimpleCupti.counterdata");
    std::string CounterDataSBFileName("SimpleCupti.counterdataSB");
    CUpti_ProfilerReplayMode profilerReplayMode = CUPTI_KernelReplay;
    CUpti_ProfilerRange profilerRange = CUPTI_AutoRange;
    int deviceCount, deviceNum;
    char* metricName;

    printf("Usage: %s [device_num] [metric_names comma separated]\n", argv[0]);

    DRIVER_API_CALL(hipInit(0));
    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        return -2;
    }

    if (argc > 1)
        deviceNum = atoi(argv[1]);
    else
        deviceNum = 0;
    printf("CUDA Device Number: %d\n", deviceNum);

    // Get the names of the metrics to collect
    if (argc > 2) {
        metricName = strtok(argv[2], ",");
        while(metricName != NULL)
        {
            metricNames.push_back(metricName);
            metricName = strtok(NULL, ",");
        }
    }
    else {
        metricNames.push_back(METRIC_NAME);
    }

    CUpti_Profiler_Initialize_Params profilerInitializeParams = {CUpti_Profiler_Initialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerInitialize(&profilerInitializeParams));
    /* Get chip name for the cuda  device */
    CUpti_Device_GetChipName_Params getChipNameParams = { CUpti_Device_GetChipName_Params_STRUCT_SIZE };
    getChipNameParams.deviceIndex = deviceNum;
    CUPTI_API_CALL(cuptiDeviceGetChipName(&getChipNameParams));
    std::string chipName(getChipNameParams.pChipName);

    /* Generate configuration for metrics, this can also be done offline*/
    NVPW_InitializeHost_Params initializeHostParams = { NVPW_InitializeHost_Params_STRUCT_SIZE };
    NVPW_API_CALL(NVPW_InitializeHost(&initializeHostParams));

    if (metricNames.size()) {
        if(!NV::Metric::Config::GetConfigImage(chipName, metricNames, configImage))
        {
            std::cout << "Failed to create configImage" << std::endl;
            exit(-1);
        }
        if(!NV::Metric::Config::GetCounterDataPrefixImage(chipName, metricNames, counterDataImagePrefix))
        {
            std::cout << "Failed to create counterDataImagePrefix" << std::endl;
            exit(-1);
        }
    }
    else
    {
        std::cout << "No metrics provided to profile" << std::endl;
        exit(-1);
    }

    if(!CreateCounterDataImage(counterDataImage, counterDataScratchBuffer, counterDataImagePrefix))
    {
        std::cout << "Failed to create counterDataImage" << std::endl;
        exit(-1);
    }

    if(!runTest(deviceNum, configImage, counterDataScratchBuffer, counterDataImage, profilerReplayMode, profilerRange))
    {
        std::cout << "Failed to run sample" << std::endl;
        exit(-1);
    }
    CUpti_Profiler_DeInitialize_Params profilerDeInitializeParams = {CUpti_Profiler_DeInitialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerDeInitialize(&profilerDeInitializeParams));

    /* Dump counterDataImage in file */
    WriteBinaryFile(CounterDataFileName.c_str(), counterDataImage);
    WriteBinaryFile(CounterDataSBFileName.c_str(), counterDataScratchBuffer);

    /* Evaluation of metrics collected in counterDataImage, this can also be done offline*/
    NV::Metric::Eval::PrintMetricValues(chipName, counterDataImage, metricNames);

    return 0;
}