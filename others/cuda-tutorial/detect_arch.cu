/**
 * CUDA Architecture Detection
 * 
 * This small utility program detects the architecture of the primary CUDA GPU
 * and outputs it in the format required for NVCC's -arch flag (e.g., sm_75).
 * 
 * Used by the Makefile to automatically set the appropriate architecture.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int device = 0;
    hipError_t error = hipSuccess;
    hipDeviceProp_t deviceProp;
    
    error = hipGetDeviceCount(&device);
    if (error != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(error));
        return 1;
    }
    
    if (device == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        return 1;
    }
    
    // Get properties of the first device
    error = hipGetDeviceProperties(&deviceProp, 0);
    if (error != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed: %s\n", hipGetErrorString(error));
        return 1;
    }
    
    // Print in the format expected by nvcc -arch flag
    printf("sm_%d%d\n", deviceProp.major, deviceProp.minor);
    
    return 0;
} 