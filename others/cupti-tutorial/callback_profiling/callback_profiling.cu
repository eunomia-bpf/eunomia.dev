#include "hip/hip_runtime.h"
//
// Copyright 2020-2022 NVIDIA Corporation. All rights reserved
//

// System headers
#include <string>
#include <stdio.h>
#include <stdlib.h>

// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUPTI headers
#include "helper_cupti.h"
#include <cupti_target.h>
#include <cupti_callbacks.h>
#include <cupti_driver_cbid.h>
#include <cupti_profiler_target.h>

// Perfworks headers
#include <nvperf_host.h>

// Make use of example code wrappers for NVPW calls.
#include <Eval.h>
#include <Metric.h>
#include <FileOp.h>

// Structures
typedef struct ProfilingData_t
{
    int numRanges = 2;
    bool bProfiling = false;
    std::string chipName;
    std::vector<std::string> metricNames;
    std::string counterDataFileName = "SimpleCupti.counterdata";
    std::string counterDataSBFileName = "SimpleCupti.counterdataSB";
    CUpti_ProfilerRange profilerRange = CUPTI_AutoRange;
    CUpti_ProfilerReplayMode profilerReplayMode = CUPTI_UserReplay;
    bool allPassesSubmitted = true;
    std::vector<uint8_t> counterDataImagePrefix;
    std::vector<uint8_t> configImage;
    std::vector<uint8_t> counterDataImage;
    std::vector<uint8_t> counterDataScratchBuffer;
} ProfilingData;

// Macros
#define METRIC_NAME "sm__ctas_launched.sum"

// Kernels
__global__ void
VectorAdd(
    const int *pA,
    const int *pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        pC[i] = pA[i] + pB[i];
    }
}

// Functions
void
EnableProfiling(
    ProfilingData *pProfilingData)
{
    CUpti_Profiler_EnableProfiling_Params enableProfilingParams = { CUpti_Profiler_EnableProfiling_Params_STRUCT_SIZE };
    if (pProfilingData->profilerReplayMode == CUPTI_KernelReplay)
    {
        CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
    }
    else if (pProfilingData->profilerReplayMode == CUPTI_UserReplay)
    {
        CUpti_Profiler_BeginPass_Params beginPassParams = { CUpti_Profiler_BeginPass_Params_STRUCT_SIZE };
        CUPTI_API_CALL(cuptiProfilerBeginPass(&beginPassParams));
        CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
    }
}

void
DisableProfiling(
    ProfilingData *pProfilingData)
{
    CUpti_Profiler_DisableProfiling_Params disableProfilingParams = { CUpti_Profiler_DisableProfiling_Params_STRUCT_SIZE };
    CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));

    if (pProfilingData->profilerReplayMode == CUPTI_UserReplay)
    {
        CUpti_Profiler_EndPass_Params endPassParams = { CUpti_Profiler_EndPass_Params_STRUCT_SIZE };
        CUPTI_API_CALL(cuptiProfilerEndPass(&endPassParams));
        pProfilingData->allPassesSubmitted = (endPassParams.allPassesSubmitted == 1) ? true : false;
    }
    else if (pProfilingData->profilerReplayMode == CUPTI_KernelReplay)
    {
        pProfilingData->allPassesSubmitted = true;
    }

    if (pProfilingData->allPassesSubmitted)
    {
        CUpti_Profiler_FlushCounterData_Params flushCounterDataParams = { CUpti_Profiler_FlushCounterData_Params_STRUCT_SIZE };
        CUPTI_API_CALL(cuptiProfilerFlushCounterData(&flushCounterDataParams));
    }
}

void
BeginSession(
    ProfilingData *pProfilingData)
{
    CUpti_Profiler_BeginSession_Params beginSessionParams = { CUpti_Profiler_BeginSession_Params_STRUCT_SIZE };
    beginSessionParams.ctx = NULL;
    beginSessionParams.counterDataImageSize = pProfilingData->counterDataImage.size();
    beginSessionParams.pCounterDataImage = &pProfilingData->counterDataImage[0];
    beginSessionParams.counterDataScratchBufferSize = pProfilingData->counterDataScratchBuffer.size();
    beginSessionParams.pCounterDataScratchBuffer = &pProfilingData->counterDataScratchBuffer[0];
    beginSessionParams.range = pProfilingData->profilerRange;
    beginSessionParams.replayMode = pProfilingData->profilerReplayMode;
    beginSessionParams.maxRangesPerPass = pProfilingData->numRanges;
    beginSessionParams.maxLaunchesPerPass = pProfilingData->numRanges;
    CUPTI_API_CALL(cuptiProfilerBeginSession(&beginSessionParams));
}

void
SetConfig(
    ProfilingData *pProfilingData)
{
    CUpti_Profiler_SetConfig_Params setConfigParams = { CUpti_Profiler_SetConfig_Params_STRUCT_SIZE };
    setConfigParams.pConfig = &pProfilingData->configImage[0];
    setConfigParams.configSize = pProfilingData->configImage.size();
    setConfigParams.passIndex = 0;
    CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));
}

void
CreateCounterDataImage(
    int numRanges,
    std::vector<uint8_t>& counterDataImagePrefix,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImage)
{
    CUpti_Profiler_CounterDataImageOptions counterDataImageOptions;
    counterDataImageOptions.pCounterDataPrefix = &counterDataImagePrefix[0];
    counterDataImageOptions.counterDataPrefixSize = counterDataImagePrefix.size();
    counterDataImageOptions.maxNumRanges = numRanges;
    counterDataImageOptions.maxNumRangeTreeNodes = numRanges;
    counterDataImageOptions.maxRangeNameLength = 64;

    CUpti_Profiler_CounterDataImage_CalculateSize_Params calculateSizeParams = { CUpti_Profiler_CounterDataImage_CalculateSize_Params_STRUCT_SIZE };
    calculateSizeParams.pOptions = &counterDataImageOptions;
    calculateSizeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateSize(&calculateSizeParams));

    CUpti_Profiler_CounterDataImage_Initialize_Params initializeParams = { CUpti_Profiler_CounterDataImage_Initialize_Params_STRUCT_SIZE };
    initializeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    initializeParams.pOptions = &counterDataImageOptions;
    initializeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    counterDataImage.resize(calculateSizeParams.counterDataImageSize);
    initializeParams.pCounterDataImage = &counterDataImage[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitialize(&initializeParams));

    CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params scratchBufferSizeParams = { CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params_STRUCT_SIZE };
    scratchBufferSizeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    scratchBufferSizeParams.pCounterDataImage = initializeParams.pCounterDataImage;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateScratchBufferSize(&scratchBufferSizeParams));
    counterDataScratchBuffer.resize(scratchBufferSizeParams.counterDataScratchBufferSize);

    CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params initScratchBufferParams = { CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params_STRUCT_SIZE };
    initScratchBufferParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    initScratchBufferParams.pCounterDataImage = initializeParams.pCounterDataImage;
    initScratchBufferParams.counterDataScratchBufferSize = scratchBufferSizeParams.counterDataScratchBufferSize;
    initScratchBufferParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitializeScratchBuffer(&initScratchBufferParams));
}

void
SetupProfiling(
    ProfilingData *pProfilingData)
{
    // Generate configuration for metrics, this can also be done offline.
    NVPW_InitializeHost_Params initializeHostParams = { NVPW_InitializeHost_Params_STRUCT_SIZE };
    NVPW_API_CALL(NVPW_InitializeHost(&initializeHostParams));

    if (pProfilingData->metricNames.size())
    {
        if (!NV::Metric::Config::GetConfigImage(pProfilingData->chipName, pProfilingData->metricNames, pProfilingData->configImage))
        {
            std::cout << "Failed to create configImage" << std::endl;
            exit(EXIT_FAILURE);
        }
        if (!NV::Metric::Config::GetCounterDataPrefixImage(pProfilingData->chipName, pProfilingData->metricNames, pProfilingData->counterDataImagePrefix))
        {
            std::cout << "Failed to create counterDataImagePrefix" << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    else
    {
        std::cout << "No metrics provided to profile" << std::endl;
        exit(EXIT_FAILURE);
    }

    CreateCounterDataImage(pProfilingData->numRanges, pProfilingData->counterDataImagePrefix,
                           pProfilingData->counterDataScratchBuffer, pProfilingData->counterDataImage);

    BeginSession(pProfilingData);
    SetConfig(pProfilingData);
}

void
StopProfiling(
    ProfilingData *pProfilingData)
{
    CUpti_Profiler_UnsetConfig_Params unsetConfigParams = { CUpti_Profiler_UnsetConfig_Params_STRUCT_SIZE };
    CUpti_Profiler_EndSession_Params endSessionParams = { CUpti_Profiler_EndSession_Params_STRUCT_SIZE };
    CUpti_Profiler_DeInitialize_Params profilerDeInitializeParams = {CUpti_Profiler_DeInitialize_Params_STRUCT_SIZE};

    CUPTI_API_CALL(cuptiProfilerUnsetConfig(&unsetConfigParams));
    CUPTI_API_CALL(cuptiProfilerEndSession(&endSessionParams));
    CUPTI_API_CALL(cuptiProfilerDeInitialize(&profilerDeInitializeParams));

    // Dump counterDataImage and counterDataScratchBuffer in file.
    WriteBinaryFile(pProfilingData->counterDataFileName.c_str(), pProfilingData->counterDataImage);
    WriteBinaryFile(pProfilingData->counterDataSBFileName.c_str(), pProfilingData->counterDataScratchBuffer);
}

void
ProfilingCallbackHandler(
    void *pUserData,
    CUpti_CallbackDomain domain,
    CUpti_CallbackId callbackId,
    void *pCallbackData)
{
    ProfilingData *pProfilingData = (ProfilingData *)(pUserData);
    const CUpti_CallbackData *pCallbackInfo = (CUpti_CallbackData *)pCallbackData;

    switch (domain)
    {
        case CUPTI_CB_DOMAIN_DRIVER_API:
        {
            switch (callbackId)
            {
                case CUPTI_DRIVER_TRACE_CBID_cuLaunchKernel:
                {
                    if (pCallbackInfo->callbackSite == CUPTI_API_ENTER)
                    {
                        EnableProfiling(pProfilingData);
                    }
                    else
                    {
                        DisableProfiling(pProfilingData);
                    }
                }
                break;
                default:
                    break;
            }
            break;
        }
        case CUPTI_CB_DOMAIN_RESOURCE:
        {
            switch (callbackId)
            {
                case CUPTI_CBID_RESOURCE_CONTEXT_CREATED:
                {
                    SetupProfiling(pProfilingData);
                    pProfilingData->bProfiling = true;
                }
                break;
                default:
                    break;
            }
            break;
        }
        default:
            break;
    }

}

void
InitializeVector(
    int *pVector,
    int N)
{
    for (int i = 0; i < N; i++)
    {
        pVector[i] = i;
    }
}

static void
CleanUp(
    int *pHostA,
    int *pHostB,
    int *pHostC,
    int *pDeviceA,
    int *pDeviceB,
    int *pDeviceC)
{
    // Free host memory.
    if (pHostA)
    {
        free(pHostA);
    }
    if (pHostB)
    {
        free(pHostB);
    }
    if (pHostC)
    {
        free(pHostC);
    }

    // Free device memory.
    if (pDeviceA)
    {
        RUNTIME_API_CALL(hipFree(pDeviceA));
    }
    if (pDeviceB)
    {
        RUNTIME_API_CALL(hipFree(pDeviceB));
    }
    if (pDeviceC)
    {
        RUNTIME_API_CALL(hipFree(pDeviceC));
    }
}

void
DoVectorAddition()
{
    int N = 50000;
    size_t size = N * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int* pHostA, * pHostB, * pHostC;
    int* pDeviceA, * pDeviceB, * pDeviceC;
    int i, sum;

    // Allocate input vectors pHostA and pHostB in host memory.
    pHostA = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostA);

    pHostB = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostB);

    pHostC = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostC);

    // Initialize input vectors
    InitializeVector(pHostA, N);
    InitializeVector(pHostB, N);
    memset(pHostC, 0, size);

    // Allocate vectors in device memory.
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceA, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceB, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceC, size));

    // Copy vectors from host memory to device memory.
    RUNTIME_API_CALL(hipMemcpy(pDeviceA, pHostA, size, hipMemcpyHostToDevice));
    RUNTIME_API_CALL(hipMemcpy(pDeviceB, pHostB, size, hipMemcpyHostToDevice));

    // Invoke kernel.
    threadsPerBlock = 256;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("Launching kernel: blocks %d, thread/block %d\n", blocksPerGrid, threadsPerBlock);

    VectorAdd <<< blocksPerGrid, threadsPerBlock >>> (pDeviceA, pDeviceB, pDeviceC, N);
    RUNTIME_API_CALL(hipGetLastError());

    // Copy result from device memory to host memory.
    // pHostC contains the result in host memory.
    RUNTIME_API_CALL(hipMemcpy(pHostC, pDeviceC, size, hipMemcpyDeviceToHost));

    // Verify result.
    for (i = 0; i < N; ++i)
    {
        sum = pHostA[i] + pHostB[i];
        if (pHostC[i] != sum)
        {
            fprintf(stderr, "Error: result verification failed\n");
            exit(EXIT_FAILURE);
        }
    }

    CleanUp(pHostA, pHostB, pHostC, pDeviceA, pDeviceB, pDeviceC);
}

int
main(
    int argc,
    char *argv[])
{
    hipDevice_t cuDevice = 0;
    int deviceCount, deviceNum = 0;
    int computeCapabilityMajor = 0, computeCapabilityMinor = 0;

    DRIVER_API_CALL(hipInit(0));
    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0)
    {
        printf("Warning: There is no device supporting CUDA.\nWaiving test.\n");
        exit(EXIT_WAIVED);
    }

    DRIVER_API_CALL(hipDeviceGet(&cuDevice, deviceNum));

    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));
    printf("Compute Capability of Device: %d.%d\n", computeCapabilityMajor, computeCapabilityMinor);

    // Initialize profiler API support and test device compatibility.
    CUpti_Profiler_Initialize_Params profilerInitializeParams = {CUpti_Profiler_Initialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerInitialize(&profilerInitializeParams));
    CUpti_Profiler_DeviceSupported_Params params = { CUpti_Profiler_DeviceSupported_Params_STRUCT_SIZE };
    params.cuDevice = deviceNum;
    params.api = CUPTI_PROFILER_RANGE_PROFILING;
    CUPTI_API_CALL(cuptiProfilerDeviceSupported(&params));

    if (params.isSupported != CUPTI_PROFILER_CONFIGURATION_SUPPORTED)
    {
        ::std::cerr << "Unable to profile on device " << deviceNum << ::std::endl;

        if (params.architecture == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice architecture is not supported" << ::std::endl;
        }

        if (params.sli == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice sli configuration is not supported" << ::std::endl;
        }

        if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice vgpu configuration is not supported" << ::std::endl;
        }
        else if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_DISABLED)
        {
            ::std::cerr << "\tdevice vgpu configuration disabled profiling support" << ::std::endl;
        }

        if (params.confidentialCompute == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice confidential compute configuration is not supported" << ::std::endl;
        }

        if (params.cmp == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tNVIDIA Crypto Mining Processors (CMP) are not supported" << ::std::endl;
        }

        if (params.wsl == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tWSL is not supported" << ::std::endl;
        }
        exit(EXIT_WAIVED);
    }

    ProfilingData *pProfilingData = new ProfilingData();
    for (int i = 1; i < argc; ++i)
    {
        char* arg = argv[i];
        if (strcmp(arg, "--help") == 0 || strcmp(arg, "-h") == 0)
        {
            printf("Usage: %s -d [device_num] -m [metric_names comma separated] -n [num of ranges] -r [kernel or user] -o [counterdata filename]\n", argv[0]);
            exit(EXIT_SUCCESS);
        }

        if (strcmp(arg, "--device") == 0 || strcmp(arg, "-d") == 0)
        {
            deviceNum = atoi(argv[i + 1]);
            printf("CUDA Device Number: %d\n", deviceNum);
            i++;
        }
        else if (strcmp(arg, "--metrics") == 0 || strcmp(arg, "-m") == 0)
        {
            char* metricName = strtok(argv[i + 1], ",");
            while (metricName != NULL)
            {
                pProfilingData->metricNames.push_back(metricName);
                metricName = strtok(NULL, ",");
            }
            i++;
        }
        else if (strcmp(arg, "--numRanges") == 0 || strcmp(arg, "-n") == 0)
        {
            int numRanges = atoi(argv[i + 1]);
            pProfilingData->numRanges = numRanges;
            i++;
        }
        else if (strcmp(arg, "--replayMode") == 0 || strcmp(arg, "-r") == 0)
        {
            std::string replayMode(argv[i + 1]);
            if (replayMode == "kernel")
            {
                pProfilingData->profilerReplayMode = CUPTI_KernelReplay;
            }
            else if (replayMode == "user")
            {
                pProfilingData->profilerReplayMode = CUPTI_UserReplay;
            }
            else
            {
                printf("Invalid --replayMode argument supported replayMode type 'kernel' or 'user'\n");
                exit(EXIT_FAILURE);
            }
            i++;
        }
        else if (strcmp(arg, "--outputCounterData") == 0 || strcmp(arg, "-o") == 0)
        {
            std::string outputCounterData(argv[i + 1]);
            pProfilingData->counterDataFileName = outputCounterData;
            pProfilingData->counterDataSBFileName = outputCounterData + "SB";
            i++;
        }
        else
        {
            printf("Error!! Invalid Arguments\n");
            printf("Usage: %s -d [device_num] -m [metric_names comma separated] -n [num of ranges] -r [kernel or user] -o [counterdata filename]\n", argv[0]);
            exit(EXIT_FAILURE);
        }
    }

    if (pProfilingData->metricNames.empty())
    {
        pProfilingData->metricNames.push_back(METRIC_NAME);
    }

    CUpti_Device_GetChipName_Params getChipNameParams = { CUpti_Device_GetChipName_Params_STRUCT_SIZE };
    getChipNameParams.deviceIndex = deviceNum;
    CUPTI_API_CALL(cuptiDeviceGetChipName(&getChipNameParams));
    pProfilingData->chipName = getChipNameParams.pChipName;

    CUpti_SubscriberHandle subscriber;
    CUPTI_API_CALL(cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)ProfilingCallbackHandler, pProfilingData));
    CUPTI_API_CALL(cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_RESOURCE, CUPTI_CBID_RESOURCE_CONTEXT_CREATED));
    CUPTI_API_CALL(cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_DRIVER_API, CUPTI_DRIVER_TRACE_CBID_cuLaunchKernel));

    do
    {
        DoVectorAddition();
    }
    while (!pProfilingData->allPassesSubmitted);

    if (pProfilingData->bProfiling)
    {
        StopProfiling(pProfilingData);
        pProfilingData->bProfiling = false;

        // Evaluation of metrics collected in counterDataImage, this can also be done offline.
        NV::Metric::Eval::PrintMetricValues(pProfilingData->chipName, pProfilingData->counterDataImage, pProfilingData->metricNames);
    }

    delete pProfilingData;

    exit(EXIT_SUCCESS);
}
