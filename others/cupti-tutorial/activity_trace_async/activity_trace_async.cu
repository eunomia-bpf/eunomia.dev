#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2022 NVIDIA Corporation. All rights reserved
 *
 * Sample CUPTI app to print a trace of CUDA API and GPU activity
 */

// System headers
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUPTI headers
#include "helper_cupti_activity.h"

// Macros
#define COMPUTE_N 50000

// Kernels
__global__ void
VectorAdd(
    const int* pA,
    const int* pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        pC[i] = pA[i] + pB[i];
    }
}

__global__ void
VectorSubtract(
    const int* pA,
    const int* pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        pC[i] = pA[i] - pB[i];
    }
}

// Functions
static void
DoPass(
    hipStream_t stream)
{
    int *pHostA, *pHostB, *pHostC;
    int *pDeviceA, *pDeviceB, *pDeviceC;
    size_t size = COMPUTE_N * sizeof(int);
    int threadsPerBlock = 256;
    int blocksPerGrid = 0;

    // Allocate input vectors pHostA and pHostB in host memory.
    // Don't bother to initialize.
    pHostA = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostA);

    pHostB = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostB);

    pHostC = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostB);


    // Allocate vectors in device memory.
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceA, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceB, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceC, size));

    RUNTIME_API_CALL(hipMemcpyAsync(pDeviceA, pHostA, size, hipMemcpyHostToDevice, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(pDeviceB, pHostB, size, hipMemcpyHostToDevice, stream));

    blocksPerGrid = (COMPUTE_N + threadsPerBlock - 1) / threadsPerBlock;

    VectorAdd <<< blocksPerGrid, threadsPerBlock, 0, stream >>> (pDeviceA, pDeviceB, pDeviceC, COMPUTE_N);
    RUNTIME_API_CALL(hipGetLastError());

    VectorSubtract <<< blocksPerGrid, threadsPerBlock, 0, stream >>> (pDeviceA, pDeviceB, pDeviceC, COMPUTE_N);
    RUNTIME_API_CALL(hipGetLastError());

    RUNTIME_API_CALL(hipMemcpyAsync(pHostC, pDeviceC, size, hipMemcpyDeviceToHost, stream));

    if (stream == 0)
    {
        RUNTIME_API_CALL(hipDeviceSynchronize());
    }
    else
    {
        RUNTIME_API_CALL(hipStreamSynchronize(stream));
    }

    // Free host memory.
    if (pHostA)
    {
        free(pHostA);
    }
    if (pHostB)
    {
        free(pHostB);
    }
    if (pHostC)
    {
        free(pHostC);
    }

    // Free device memory.
    if (pDeviceA)
    {
        RUNTIME_API_CALL(hipFree(pDeviceA));
    }
    if (pDeviceB)
    {
        RUNTIME_API_CALL(hipFree(pDeviceB));
    }
    if (pDeviceC)
    {
        RUNTIME_API_CALL(hipFree(pDeviceC));
    }
}

static void
SetupCupti()
{
    UserData *pUserData = (UserData *)malloc(sizeof(UserData));
    MEMORY_ALLOCATION_CALL(pUserData);

    memset(pUserData, 0, sizeof(UserData));
    pUserData->pPostProcessActivityRecords = NULL;
    pUserData->printActivityRecords        = 1;

    // Common CUPTI Initialization
    InitCuptiTrace(pUserData, NULL, stdout);

    // Device activity record is created when CUDA initializes, so we
    // want to enable it before hipInit() or any CUDA runtime call.
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_DEVICE));
    // Enable all other activity record kinds.
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_CONTEXT));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_DRIVER));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_RUNTIME));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_MEMCPY));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_MEMSET));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_NAME));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_MARKER));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_OVERHEAD));
}

int
main(
    int argc,
    char *argv[])
{
    hipDevice_t device;
    char deviceName[256];
    int deviceId = 0, deviceCount = 0;

    SetupCupti();

    // Intialize CUDA
    DRIVER_API_CALL(hipInit(0));

    RUNTIME_API_CALL(hipGetDeviceCount(&deviceCount));

    for (deviceId = 0; deviceId < deviceCount; deviceId++)
    {
        DRIVER_API_CALL(hipDeviceGet(&device, deviceId));
        DRIVER_API_CALL(hipDeviceGetName(deviceName, 256, device));
        printf("Device Name: %s\n", deviceName);

        RUNTIME_API_CALL(hipSetDevice(deviceId));

        // DoPass with default stream
        DoPass(0);

        // DoPass with user stream
        hipStream_t stream;
        RUNTIME_API_CALL(hipStreamCreate(&stream));
        DoPass(stream);

        RUNTIME_API_CALL(hipDeviceSynchronize());

        RUNTIME_API_CALL(hipDeviceReset());
    }

    DeInitCuptiTrace();

    exit(EXIT_SUCCESS);
}
