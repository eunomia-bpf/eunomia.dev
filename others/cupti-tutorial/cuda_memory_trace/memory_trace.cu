#include "hip/hip_runtime.h"
/*
 * Copyright 2021-2022 NVIDIA Corporation. All rights reserved
 *
 * Sample CUPTI app to print trace of CUDA memory operations.
 * The sample also traces CUDA memory operations done via
 * default memory pool.
 *
 */

// System headers
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUPTI headers
#include "helper_cupti_activity.h"

// Kernels
__global__ void
VectorAdd(
    const float *pA,
    const float *pB,
    float *pC,
    int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        pC[i] = pA[i] + pB[i];
    }
}

// Functions
static void
DoMemoryAllocations()
{
    hipDeviceProp_t deviceProperties;
    RUNTIME_API_CALL(hipGetDeviceProperties(&deviceProperties, 0));

    if (!deviceProperties.managedMemory)
    {
        // This samples requires being run on a device that supports Unified Memory.
        printf("Warning: Unified Memory not supported on this device. Waiving sample.\n");
        return;
    }

    int nElements = 1048576;
    size_t size = nElements * sizeof(int);

    int *pHostA, *pHostB;
    int *pDeviceA, *pDeviceB;

    // Allocate memory.
    RUNTIME_API_CALL(hipHostMalloc((void **)&pHostA, size));
    RUNTIME_API_CALL(hipHostAlloc((void **)&pHostB, size, hipHostMallocPortable));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceA, size));
    RUNTIME_API_CALL(hipMallocManaged((void **)&pDeviceB, size, hipMemAttachGlobal));

    // Free the allocated memory.
    RUNTIME_API_CALL(hipHostFree(pHostA));
    RUNTIME_API_CALL(hipHostFree(pHostB));
    RUNTIME_API_CALL(hipFree(pDeviceA));
    RUNTIME_API_CALL(hipFree(pDeviceB));
}

static void
DoMemoryAllocationsViaMemoryPool()
{
    int nElements = 1048576;
    size_t bytes = nElements * sizeof(float);

    float *pHostA, *pHostB, *pHostC;
    float *pDeviceA, *pDeviceB, *pDeviceC;
    hipStream_t stream;

    int isMemPoolSupported = 0;
    hipError_t cudaStatus = hipSuccess;
    cudaStatus = hipDeviceGetAttribute(&isMemPoolSupported, hipDeviceAttributeMemoryPoolsSupported, 0);
    // For enhance compatibility cases, the attribute hipDeviceAttributeMemoryPoolsSupported might not be present.
    // return early if Runtime API does not return hipSuccess.
    if (!isMemPoolSupported || cudaStatus != hipSuccess)
    {
        printf("Warning: Memory pool not supported on this device. Waiving sample.\n");
        return;
    }

    // Allocate and initialize memory on host and device.
    pHostA = (float*) malloc(bytes);
    MEMORY_ALLOCATION_CALL(pHostA);

    pHostB = (float*) malloc(bytes);
    MEMORY_ALLOCATION_CALL(pHostB);

    pHostC = (float*) malloc(bytes);
    MEMORY_ALLOCATION_CALL(pHostC);

    for (int n = 0; n < nElements; n++)
    {
        pHostA[n] = rand() / (float)RAND_MAX;
        pHostB[n] = rand() / (float)RAND_MAX;
    }

    RUNTIME_API_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // Allocate memory using default memory pool.
    RUNTIME_API_CALL(hipMallocAsync(&pDeviceA, bytes, stream));
    RUNTIME_API_CALL(hipMallocAsync(&pDeviceB, bytes, stream));
    RUNTIME_API_CALL(hipMallocAsync(&pDeviceC, bytes, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(pDeviceA, pHostA, bytes, hipMemcpyHostToDevice, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(pDeviceB, pHostB, bytes, hipMemcpyHostToDevice, stream));

    dim3 block(256);
    dim3 grid((unsigned int)ceil(nElements/(float)block.x));
    VectorAdd<<<grid, block, 0, stream>>>(pDeviceA, pDeviceB, pDeviceC, nElements);

    // Free the allocated memory.
    RUNTIME_API_CALL(hipFreeAsync(pDeviceA, stream));
    RUNTIME_API_CALL(hipFreeAsync(pDeviceB, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(pHostC, pDeviceC, bytes, hipMemcpyDeviceToHost, stream));

    RUNTIME_API_CALL(hipStreamSynchronize(stream));
    RUNTIME_API_CALL(hipFree(pDeviceC));

    RUNTIME_API_CALL(hipStreamDestroy(stream));

    // Free host memory.
    if (pHostA)
    {
        free(pHostA);
    }
    if (pHostB)
    {
        free(pHostB);
    }
    if (pHostC)
    {
        free(pHostC);
    }
}

static void
SetupCupti()
{
    UserData *pUserData = (UserData *)malloc(sizeof(UserData));
    MEMORY_ALLOCATION_CALL(pUserData);

    memset(pUserData, 0, sizeof(UserData));
    pUserData->pPostProcessActivityRecords = NULL;
    pUserData->printActivityRecords        = 1;

    // Common CUPTI Initialization
    InitCuptiTrace(pUserData, NULL, stdout);

    // Enable CUPTI activities related to memory allocation
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_MEMORY2));
    CUPTI_API_CALL_VERBOSE(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_MEMORY_POOL));

}

int
main(
    int argc,
    char *argv[])
{
    SetupCupti();

    // Intialize CUDA.
    DRIVER_API_CALL(hipInit(0));

    char deviceName[256];
    hipDevice_t device;
    DRIVER_API_CALL(hipDeviceGet(&device, 0));
    DRIVER_API_CALL(hipDeviceGetName(deviceName, 256, device));
    printf("Device Name: %s\n", deviceName);
    RUNTIME_API_CALL(hipSetDevice(0));

    DoMemoryAllocations();
    DoMemoryAllocationsViaMemoryPool();

    DeInitCuptiTrace();

    exit(EXIT_SUCCESS);
}
