#include "hip/hip_runtime.h"
/*
 * Copyright 2014-2022 NVIDIA Corporation. All rights reserved
 *
 * Sample CUPTI app to demonstrate the usage of pc sampling.
 * This app will work on devices with compute capability 5.2
 * or 6.0 and higher.
 */

// System headers
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUPTI headers
#include "helper_cupti_activity.h"

// Macros
#define ARRAY_SIZE 32
#define THREADS_PER_BLOCK 32

// Kernels
__global__ void
VectorAdd(
    const int *pA,
    const int *pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        pC[i] = pA[i] + pB[i];
    }
}

// Functions
static void
DoPass(
    hipStream_t stream)
{
    int *pHostA, *pHostB, *pHostC;
    int *pDeviceA, *pDeviceB, *pDeviceC;
    size_t size = ARRAY_SIZE * sizeof(int);
    int blocksPerGrid = 0;

    hipCtx_t context;

    // Allocate input vectors pHostA and pHostB in host memory.
    pHostA = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostA);

    pHostB = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostB);

    pHostC = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostC);

    // Allocate vectors in device memory.
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceA, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceB, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceC, size));

    CUpti_ActivityPCSamplingConfig configPC;
    configPC.size = sizeof(CUpti_ActivityPCSamplingConfig);
    configPC.samplingPeriod=CUPTI_ACTIVITY_PC_SAMPLING_PERIOD_MIN;
    configPC.samplingPeriod2 = 0;
    hipCtxGetCurrent(&context);

    // Configure api needs to be called before activity enable for chips till Pascal chips.
    // Order does not matter for Volta+ chips.
    CUPTI_API_CALL(cuptiActivityConfigurePCSampling(context, &configPC));
    CUPTI_API_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_PC_SAMPLING));

    RUNTIME_API_CALL(hipMemcpyAsync(pDeviceA, pHostA, size, hipMemcpyHostToDevice, stream));
    RUNTIME_API_CALL(hipMemcpyAsync(pDeviceB, pHostB, size, hipMemcpyHostToDevice, stream));

    blocksPerGrid = (ARRAY_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    VectorAdd <<< blocksPerGrid, THREADS_PER_BLOCK, 0, stream >>> (pDeviceA, pDeviceB, pDeviceC, ARRAY_SIZE);
    RUNTIME_API_CALL(hipGetLastError());

    RUNTIME_API_CALL(hipMemcpyAsync(pHostC, pDeviceC, size, hipMemcpyDeviceToHost, stream));

    if (stream == 0)
    {
        RUNTIME_API_CALL(hipDeviceSynchronize());
    }
    else
    {
        RUNTIME_API_CALL(hipStreamSynchronize(stream));
    }

    // Free host memory.
    if (pHostA)
    {
        free(pHostA);
    }
    if (pHostB)
    {
        free(pHostB);
    }
    if (pHostC)
    {
        free(pHostC);
    }

    // Free device memory.
    if (pDeviceA)
    {
        RUNTIME_API_CALL(hipFree(pDeviceA));
    }
    if (pDeviceB)
    {
        RUNTIME_API_CALL(hipFree(pDeviceB));
    }
    if (pDeviceC)
    {
        RUNTIME_API_CALL(hipFree(pDeviceC));
    }
}

static void
SetupCupti()
{
    UserData *pUserData = (UserData *)malloc(sizeof(UserData));
    MEMORY_ALLOCATION_CALL(pUserData);

    memset(pUserData, 0, sizeof(UserData));
    pUserData->pPostProcessActivityRecords = NULL;
    pUserData->printActivityRecords        = 1;

    // Common CUPTI Initialization.
    InitCuptiTrace(pUserData, NULL, stdout);
}

int
main(
    int argc,
    char *argv[])
{
    int deviceNum = 0;
    hipDeviceProp_t prop;

    printf("Warning: The PC Sampling Activity API from the cupti_activity.h header is deprecated on Volta+ GPU architectures\n"
           "and will be removed in a future release. Additionally, this API is not supported on Blackwell+ GPU architectures.\n"
           "It is recommended to transition to the PC Sampling API from the cupti_pcsampling.h header, which is supported\n"
           "on Volta+ GPU architectures.\n\n");


    SetupCupti();

    RUNTIME_API_CALL(hipGetDevice(&deviceNum));
    RUNTIME_API_CALL(hipGetDeviceProperties(&prop, deviceNum));
    printf("Device Name: %s\n", prop.name);
    printf("Device compute capability: %d.%d\n", prop.major, prop.minor);

    if (!((prop.major > 5) || ((prop.major == 5) && (prop.minor == 2)) || prop.minor <= 9))
    {
        printf("Warning: Sample is waived on this device.\nPC Sampling Activity API is supported on devices with compute capability 5.2 and from 6.0 to 9.0.\n");
        exit(EXIT_WAIVED);
    }

    DoPass(0);

    RUNTIME_API_CALL(hipDeviceSynchronize());
    RUNTIME_API_CALL(hipDeviceReset());

    DeInitCuptiTrace();

    exit(EXIT_SUCCESS);
}
