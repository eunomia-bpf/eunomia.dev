/*
 * Copyright 2011-2015 NVIDIA Corporation. All rights reserved
 *
 * Sample CUPTI app to print a trace of CUDA API and GPU activity
 */ 
#include <hip/hip_runtime.h>

#include <cupti.h>
#include <stdio.h>
#include <string.h>

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define COMPUTE_N 50000

extern void initTrace(void);
extern void finiTrace(void);

// Kernels
__global__ void 
VecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

__global__ void 
VecSub(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] - B[i];
}

static void
do_pass(hipStream_t stream)
{
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;
  size_t size = COMPUTE_N * sizeof(int);
  int threadsPerBlock = 256;
  int blocksPerGrid = 0;
  
  // Allocate input vectors h_A and h_B in host memory
  // don't bother to initialize
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);
  
  // Allocate vectors in device memory
  RUNTIME_API_CALL(hipMalloc((void**)&d_A, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_B, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_C, size));

  RUNTIME_API_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream));
  RUNTIME_API_CALL(hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream));

  blocksPerGrid = (COMPUTE_N + threadsPerBlock - 1) / threadsPerBlock;
  VecAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, COMPUTE_N);
  VecSub<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, COMPUTE_N);
    
  RUNTIME_API_CALL(hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream));

  if (stream == 0)
    RUNTIME_API_CALL(hipDeviceSynchronize());
  else
    RUNTIME_API_CALL(hipStreamSynchronize(stream));

  free(h_A);
  free(h_B);
  free(h_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

int
main(int argc, char *argv[])
{
  hipDevice_t device;  
  char deviceName[32];
  int deviceNum = 0, devCount = 0;

  // initialize the activity trace
  // make sure activity is enabled before any CUDA API
  initTrace();

  DRIVER_API_CALL(hipInit(0));
  
  RUNTIME_API_CALL(hipGetDeviceCount(&devCount));
  for (deviceNum=0; deviceNum<devCount; deviceNum++) {
      DRIVER_API_CALL(hipDeviceGet(&device, deviceNum));
      DRIVER_API_CALL(hipDeviceGetName(deviceName, 32, device));
      printf("Device Name: %s\n", deviceName);

      RUNTIME_API_CALL(hipSetDevice(deviceNum));
      // do pass default stream
      do_pass(0);

      // do pass with user stream
      hipStream_t stream0;
      RUNTIME_API_CALL(hipStreamCreate(&stream0));
      do_pass(stream0);

      hipDeviceSynchronize();

      // Flush all remaining CUPTI buffers before resetting the device.
      // This can also be called in the cudaDeviceReset callback.
      cuptiActivityFlushAll(0);

      hipDeviceReset();
  }
  return 0;
}

