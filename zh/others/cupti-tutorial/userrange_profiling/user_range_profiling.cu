#include "hip/hip_runtime.h"
// Copyright 2021-2022 NVIDIA Corporation. All rights reserved

// Standard STL headers
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string>

// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

// CUPTI headers
#include "helper_cupti.h"
#include <cupti_target.h>
#include <cupti_profiler_target.h>

// NVPW headers
#include <nvperf_host.h>

// Make use of example code wrappers for NVPW calls
#include <Eval.h>
#include <Metric.h>
#include <Utils.h>
#include <FileOp.h>

// Global Variables
static int s_NumRanges = 1;

// Macros
#define METRIC_NAME "smsp__warps_launched.avg"

// Kernels
__global__ void
VectorAdd(
    const int *pA,
    const int *pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        pC[i] = pA[i] + pB[i];
    }
}

__global__ void
VectorSubtract(
    const int *pA,
    const int *pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
       pC[i] = pA[i] - pB[i];
    }
}


static void
InitializeVector(
    int *pVector,
    int N)
{
    for (int i = 0; i < N; i++)
    {
        pVector[i] = i;
    }
}

static void
CleanUp(
    int *pHostA,
    int *pHostB,
    int *pHostC,
    int *pHostD,
    int *pDeviceA,
    int *pDeviceB,
    int *pDeviceC,
    int *pDeviceD)
{
    // Free host memory.
    if (pHostA)
    {
        free(pHostA);
    }
    if (pHostB)
    {
        free(pHostB);
    }
    if (pHostC)
    {
        free(pHostC);
    }
    if (pHostD)
    {
        free(pHostD);
    }

    // Free device memory.
    if (pDeviceA)
    {
        RUNTIME_API_CALL(hipFree(pDeviceA));
    }
    if (pDeviceB)
    {
        RUNTIME_API_CALL(hipFree(pDeviceB));
    }
    if (pDeviceC)
    {
        RUNTIME_API_CALL(hipFree(pDeviceC));
    }
    if (pDeviceD)
    {
        RUNTIME_API_CALL(hipFree(pDeviceD));
    }
}

static void
DoVectorAddSubtract()
{
    int N = 50000;
    size_t size = N * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int *pHostA, *pHostB, *pHostC, *pHostD;
    int *pDeviceA, *pDeviceB, *pDeviceC, *pDeviceD;
    int i, sum, diff;

    // Allocate input vectors pHostA and pHostB in host memory.
    pHostA = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostA);

    pHostB = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostB);

    pHostC = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostC);

    pHostD = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostD);

    // Initialize input vectors.
    InitializeVector(pHostA, N);
    InitializeVector(pHostB, N);
    memset(pHostC, 0, size);
    memset(pHostD, 0, size);

    // Allocate vectors in device memory.
    RUNTIME_API_CALL(hipMalloc((void**)&pDeviceA, size));
    RUNTIME_API_CALL(hipMalloc((void**)&pDeviceB, size));
    RUNTIME_API_CALL(hipMalloc((void**)&pDeviceC, size));
    RUNTIME_API_CALL(hipMalloc((void**)&pDeviceD, size));

    // Copy vectors from host memory to device memory.
    RUNTIME_API_CALL(hipMemcpy(pDeviceA, pHostA, size, hipMemcpyHostToDevice));
    RUNTIME_API_CALL(hipMemcpy(pDeviceB, pHostB, size, hipMemcpyHostToDevice));

    // Invoke kernel
    threadsPerBlock = 256;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("Launching kernel: blocks %d, thread/block %d\n", blocksPerGrid, threadsPerBlock);

    VectorAdd <<< blocksPerGrid, threadsPerBlock >>> (pDeviceA, pDeviceB, pDeviceC, N);
    RUNTIME_API_CALL(hipGetLastError());

    VectorSubtract <<< blocksPerGrid, threadsPerBlock >>> (pDeviceA, pDeviceB, pDeviceD, N);
    RUNTIME_API_CALL(hipGetLastError());

    // Copy result from device memory to host memory.
    // pHostC contains the result in host memory.
    RUNTIME_API_CALL(hipMemcpy(pHostC, pDeviceC, size, hipMemcpyDeviceToHost));
    RUNTIME_API_CALL(hipMemcpy(pHostD, pDeviceD, size, hipMemcpyDeviceToHost));

    // Verify result.
    for (i = 0; i < N; ++i)
    {
        sum  = pHostA[i] + pHostB[i];
        diff = pHostA[i] - pHostB[i];
        if (pHostC[i] != sum || pHostD[i] != diff)
        {
            fprintf(stderr, "Error: Result verification failed.\n");
            exit(EXIT_FAILURE);
        }
    }

    CleanUp(pHostA, pHostB, pHostC, pHostD, pDeviceA, pDeviceB, pDeviceC, pDeviceD);
}

bool CreateCounterDataImage(
    std::vector<uint8_t>& counterDataImage,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImagePrefix)
{
    CUpti_Profiler_CounterDataImageOptions counterDataImageOptions;
    counterDataImageOptions.pCounterDataPrefix = &counterDataImagePrefix[0];
    counterDataImageOptions.counterDataPrefixSize = counterDataImagePrefix.size();
    counterDataImageOptions.maxNumRanges = s_NumRanges;
    counterDataImageOptions.maxNumRangeTreeNodes = s_NumRanges;
    counterDataImageOptions.maxRangeNameLength = 64;

    CUpti_Profiler_CounterDataImage_CalculateSize_Params calculateSizeParams = {CUpti_Profiler_CounterDataImage_CalculateSize_Params_STRUCT_SIZE};

    calculateSizeParams.pOptions = &counterDataImageOptions;
    calculateSizeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;

    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateSize(&calculateSizeParams));

    CUpti_Profiler_CounterDataImage_Initialize_Params initializeParams = {CUpti_Profiler_CounterDataImage_Initialize_Params_STRUCT_SIZE};
    initializeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    initializeParams.pOptions = &counterDataImageOptions;
    initializeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;

    counterDataImage.resize(calculateSizeParams.counterDataImageSize);
    initializeParams.pCounterDataImage = &counterDataImage[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitialize(&initializeParams));

    CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params scratchBufferSizeParams = {CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params_STRUCT_SIZE};
    scratchBufferSizeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    scratchBufferSizeParams.pCounterDataImage = initializeParams.pCounterDataImage;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateScratchBufferSize(&scratchBufferSizeParams));

    counterDataScratchBuffer.resize(scratchBufferSizeParams.counterDataScratchBufferSize);

    CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params initScratchBufferParams = {CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params_STRUCT_SIZE};
    initScratchBufferParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;

    initScratchBufferParams.pCounterDataImage = initializeParams.pCounterDataImage;
    initScratchBufferParams.counterDataScratchBufferSize = scratchBufferSizeParams.counterDataScratchBufferSize;
    initScratchBufferParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];

    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitializeScratchBuffer(&initScratchBufferParams));

    return true;
}

bool
RunTest(
    std::vector<uint8_t>& configImage,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImage,
    CUpti_ProfilerReplayMode profilerReplayMode,
    CUpti_ProfilerRange profilerRange)
{
    hipCtx_t cuContext;
    DRIVER_API_CALL(hipCtxGetCurrent(&cuContext));

    CUpti_Profiler_BeginSession_Params beginSessionParams = {CUpti_Profiler_BeginSession_Params_STRUCT_SIZE};
    CUpti_Profiler_SetConfig_Params setConfigParams = {CUpti_Profiler_SetConfig_Params_STRUCT_SIZE};
    CUpti_Profiler_EnableProfiling_Params enableProfilingParams = {CUpti_Profiler_EnableProfiling_Params_STRUCT_SIZE};
    CUpti_Profiler_DisableProfiling_Params disableProfilingParams = {CUpti_Profiler_DisableProfiling_Params_STRUCT_SIZE};
    CUpti_Profiler_PushRange_Params pushRangeParams = {CUpti_Profiler_PushRange_Params_STRUCT_SIZE};
    CUpti_Profiler_PopRange_Params popRangeParams = {CUpti_Profiler_PopRange_Params_STRUCT_SIZE};

    beginSessionParams.ctx = NULL;
    beginSessionParams.counterDataImageSize = counterDataImage.size();
    beginSessionParams.pCounterDataImage = &counterDataImage[0];
    beginSessionParams.counterDataScratchBufferSize = counterDataScratchBuffer.size();
    beginSessionParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];
    beginSessionParams.range = profilerRange;
    beginSessionParams.replayMode = profilerReplayMode;
    beginSessionParams.maxRangesPerPass = s_NumRanges;
    beginSessionParams.maxLaunchesPerPass = s_NumRanges;

    CUPTI_API_CALL(cuptiProfilerBeginSession(&beginSessionParams));

    setConfigParams.pConfig = &configImage[0];
    setConfigParams.configSize = configImage.size();

    setConfigParams.passIndex = 0;
    setConfigParams.minNestingLevel = 1;
    setConfigParams.numNestingLevels = 1;
    CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));

    // User takes the resposiblity of replaying the kernel launches.
    CUpti_Profiler_BeginPass_Params beginPassParams = {CUpti_Profiler_BeginPass_Params_STRUCT_SIZE};
    CUpti_Profiler_EndPass_Params endPassParams = {CUpti_Profiler_EndPass_Params_STRUCT_SIZE};
    do
    {
        CUPTI_API_CALL(cuptiProfilerBeginPass(&beginPassParams));
        {
            CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
            std::string rangeName = "userrangeA";
            pushRangeParams.pRangeName = rangeName.c_str();
            CUPTI_API_CALL(cuptiProfilerPushRange(&pushRangeParams));
            {
                DoVectorAddSubtract();
            }
            CUPTI_API_CALL(cuptiProfilerPopRange(&popRangeParams));
            CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));
        }
        CUPTI_API_CALL(cuptiProfilerEndPass(&endPassParams));
    }
    while (!endPassParams.allPassesSubmitted);

    CUpti_Profiler_FlushCounterData_Params flushCounterDataParams = {CUpti_Profiler_FlushCounterData_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerFlushCounterData(&flushCounterDataParams));

    CUpti_Profiler_UnsetConfig_Params unsetConfigParams = {CUpti_Profiler_UnsetConfig_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerUnsetConfig(&unsetConfigParams));

    CUpti_Profiler_EndSession_Params endSessionParams = {CUpti_Profiler_EndSession_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerEndSession(&endSessionParams));

    return true;
}

int
main(
    int argc,
    char* argv[])
{
    hipDevice_t cuDevice;
    std::vector<std::string> metricNames;
    std::vector<uint8_t> counterDataImagePrefix;
    std::vector<uint8_t> configImage;
    std::vector<uint8_t> counterDataImage;
    std::vector<uint8_t> counterDataScratchBuffer;
    std::vector<uint8_t> counterAvailabilityImage;
    std::string CounterDataFileName("SimpleCupti.counterdata");
    std::string CounterDataSBFileName("SimpleCupti.counterdataSB");
    CUpti_ProfilerReplayMode profilerReplayMode = CUPTI_UserReplay;
    CUpti_ProfilerRange profilerRange = CUPTI_UserRange;
    int deviceCount, deviceNum;
    int computeCapabilityMajor = 0, computeCapabilityMinor = 0;
    char *pMetricName;

    printf("Usage: %s [device_num] [metric_names comma separated]\n", argv[0]);

    DRIVER_API_CALL(hipInit(0));
    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        printf("There is no device supporting CUDA.\n");
        exit(EXIT_WAIVED);
    }

    if (argc > 1)
        deviceNum = atoi(argv[1]);
    else
        deviceNum = 0;
    printf("CUDA Device Number: %d\n", deviceNum);

    DRIVER_API_CALL(hipDeviceGet(&cuDevice, deviceNum));

    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));

    printf("Compute Capability of Device: %d.%d\n", computeCapabilityMajor,computeCapabilityMinor);

    // Initialize profiler API and test device compatibility.
    CUpti_Profiler_Initialize_Params profilerInitializeParams = {CUpti_Profiler_Initialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerInitialize(&profilerInitializeParams));
    CUpti_Profiler_DeviceSupported_Params params = { CUpti_Profiler_DeviceSupported_Params_STRUCT_SIZE };
    params.cuDevice = deviceNum;
    params.api = CUPTI_PROFILER_RANGE_PROFILING;
    CUPTI_API_CALL(cuptiProfilerDeviceSupported(&params));

    if (params.isSupported != CUPTI_PROFILER_CONFIGURATION_SUPPORTED)
    {
        ::std::cerr << "Unable to profile on device " << deviceNum << ::std::endl;

        if (params.architecture == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice architecture is not supported" << ::std::endl;
        }

        if (params.sli == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice sli configuration is not supported" << ::std::endl;
        }

        if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice vgpu configuration is not supported" << ::std::endl;
        }
        else if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_DISABLED)
        {
            ::std::cerr << "\tdevice vgpu configuration disabled profiling support" << ::std::endl;
        }

        if (params.confidentialCompute == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tdevice confidential compute configuration is not supported" << ::std::endl;
        }

        if (params.cmp == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tNVIDIA Crypto Mining Processors (CMP) are not supported" << ::std::endl;
        }

        if (params.wsl == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
        {
            ::std::cerr << "\tWSL is not supported" << ::std::endl;
        }
        exit(EXIT_WAIVED);
    }

    // Get the names of the metrics to collect.
    if (argc > 2)
    {
        pMetricName = strtok(argv[2], ",");
        while(pMetricName != NULL)
        {
            metricNames.push_back(pMetricName);
            pMetricName = strtok(NULL, ",");
        }
    }
    else
    {
        metricNames.push_back(METRIC_NAME);
    }

    hipCtx_t cuContext;
    DRIVER_API_CALL(hipCtxCreate(&cuContext, 0, cuDevice));

    // Get chip name for the cuda  device.
    CUpti_Device_GetChipName_Params getChipNameParams = { CUpti_Device_GetChipName_Params_STRUCT_SIZE };
    getChipNameParams.deviceIndex = deviceNum;
    CUPTI_API_CALL(cuptiDeviceGetChipName(&getChipNameParams));
    std::string chipName(getChipNameParams.pChipName);

    CUpti_Profiler_GetCounterAvailability_Params getCounterAvailabilityParams = {CUpti_Profiler_GetCounterAvailability_Params_STRUCT_SIZE};
    getCounterAvailabilityParams.ctx = cuContext;
    CUPTI_API_CALL(cuptiProfilerGetCounterAvailability(&getCounterAvailabilityParams));

    counterAvailabilityImage.clear();
    counterAvailabilityImage.resize(getCounterAvailabilityParams.counterAvailabilityImageSize);
    getCounterAvailabilityParams.pCounterAvailabilityImage = counterAvailabilityImage.data();
    CUPTI_API_CALL(cuptiProfilerGetCounterAvailability(&getCounterAvailabilityParams));

    // Generate configuration for metrics, this can also be done offline.
    NVPW_InitializeHost_Params initializeHostParams = { NVPW_InitializeHost_Params_STRUCT_SIZE };
    NVPW_API_CALL(NVPW_InitializeHost(&initializeHostParams));

    if (metricNames.size())
    {
        if (!NV::Metric::Config::GetConfigImage(chipName, metricNames, configImage, counterAvailabilityImage.data()))
        {
            std::cout << "Failed to create configImage" << std::endl;
            exit(EXIT_FAILURE);
        }
        if (!NV::Metric::Config::GetCounterDataPrefixImage(chipName, metricNames, counterDataImagePrefix))
        {
            std::cout << "Failed to create counterDataImagePrefix" << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    else
    {
        std::cout << "No metrics provided to profile" << std::endl;
        exit(EXIT_FAILURE);
    }

    if (!CreateCounterDataImage(counterDataImage, counterDataScratchBuffer, counterDataImagePrefix))
    {
        std::cout << "Failed to create counterDataImage" << std::endl;
        exit(EXIT_FAILURE);
    }

    if (!RunTest(configImage, counterDataScratchBuffer, counterDataImage, profilerReplayMode, profilerRange))
    {
        std::cout << "Failed to run sample" << std::endl;
        exit(EXIT_FAILURE);
    }
    CUpti_Profiler_DeInitialize_Params profilerDeInitializeParams = {CUpti_Profiler_DeInitialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerDeInitialize(&profilerDeInitializeParams));

    DRIVER_API_CALL(hipCtxDestroy(cuContext));

    // Dump counterDataImage in file.
    WriteBinaryFile(CounterDataFileName.c_str(), counterDataImage);
    WriteBinaryFile(CounterDataSBFileName.c_str(), counterDataScratchBuffer);

    // Evaluation of metrics collected in counterDataImage, this can also be done offline.
    NV::Metric::Eval::PrintMetricValues(chipName, counterDataImage, metricNames);

    exit(EXIT_SUCCESS);
}
