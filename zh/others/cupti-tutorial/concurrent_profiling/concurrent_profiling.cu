#include "hip/hip_runtime.h"
// Copyright 2021 NVIDIA Corporation. All rights reserved
//
// This sample demonstrates two ways to use the CUPTI Profiler API with concurrent kernels.
// By taking the ratio of runtimes for a consecutive series of kernels, compared
// to a series of concurrent kernels, one can difinitively demonstrate that concurrent
// kernels were running while metrics were gathered and the User Replay mechanism was in use.
//
// Example:
// 4 kernel launches, with 1x, 2x, 3x, and 4x amounts of work, each sized to one SM (one warp
// of threads, one thread block).
// When run synchronously, this comes to 10x amount of work.
// When run concurrently, the longest (4x) kernel should be the only measured time (it hides the others).
// Thus w/ 4 kernels, the concurrent : consecutive time ratio should be 4:10.
// On test hardware this does simplify to 3.998:10.  As the test is affected by memory layout, this may not
// hold for certain architectures where, for example, cache sizes may optimize certain kernel calls.
//
// After demonstrating concurrency using multpile streams, this then demonstrates using multiple devices.
// In this 3rd configuration, the same concurrent workload with streams is then duplicated and run
// on each device concurrently using streams.
// In this case, the wallclock time to launch, run, and join the threads should be roughly the same as the
// wallclock time to run the single device case.  If concurrency was not working, the wallcock time
// would be (num devices) times the single device concurrent case.
//
//  * If the multiple devices have different performance, the runtime may be significantly different between
//    devices, but this does not mean concurrent profiling is not happening.

// Standard STL headers
#include <stdlib.h>
#include <chrono>
#include <cstdint>
#include <iostream>
using ::std::cerr;
using ::std::cout;
using ::std::endl;

#include <string>
using ::std::string;

#include <thread>
using ::std::thread;

#include <vector>
using ::std::vector;

// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

// CUPTI headers
#include "helper_cupti.h"
#include <cupti_target.h>
#include <cupti_profiler_target.h>

// NVPW headers
#include <nvperf_host.h>

// Make use of example code wrappers for NVPW calls
#include <Eval.h>
using ::NV::Metric::Eval::PrintMetricValues;

#include <Metric.h>
using ::NV::Metric::Config::GetConfigImage;
using ::NV::Metric::Config::GetCounterDataPrefixImage;

#include <Utils.h>
using ::NV::Metric::Utils::GetNVPWResultString;

// Consolidate CUPTI Profiler options into one location
// This contains fields for multiple levels of Profiler API configuration, with only one Session and Config
// More complicated configurations can configure multiple sessions and multiple configs w/in a session:
// Session
//     Config
//     ...
//     Config
// ...
// Session
//     Config
//     ...
//     Config

// Global  structures and variables
typedef struct ProfilingConfig_st
{
    // String name of target compute device, needed for NVPW calls.
    char const *pChipName;
    // Compute device number.
    int device;
    // Maximum number of kernel launches in any Pass in this Session.
    int maxLaunchesPerPass;
    // Maximum number of Ranges that may be encountered in this Session. (Nested Ranges are multiplicative.)
    int maxNumRanges;
    // Maximum string length for any Range in this Session.
    int maxRangeNameLength;
    // Maximum number of Ranges in any Pass in this Session.
    int maxRangesPerPass;
    // Minimum level to tag a Range within this session, must be >= 1.
    int minNestingLevels;
    // Maximum level for nested Ranges within this Session, must be >= 1.
    int numNestingLevels;
    // CUPTI_AutoRange or CUPTI_UserRange.
    CUpti_ProfilerRange rangeMode;
    // CUPTI_KernelReplay, CUPTI_UserReplay, or CUPTI_ApplicationReplay.
    CUpti_ProfilerReplayMode replayMode;
    // CUDA driver context, or NULL if default context has already been initialized.
    hipCtx_t context;
} ProfilingConfig;

// Per-device configuration, buffers, stream and device information, and device pointers.
typedef struct PerDeviceData_st
{
    int deviceID;
    // Each device (or each context) needs its own CUPTI profiling config.
    ProfilingConfig config;
     // Profiling data images.
    vector<uint8_t> counterDataImage;
    vector<uint8_t> counterDataPrefixImage;
    vector<uint8_t> counterDataScratchBufferImage;
    vector<uint8_t> configImage;
    // Each device needs its own streams.
    vector<hipStream_t> streams;
    // Device memory allocations.
    vector<double *> d_x;
    vector<double *> d_y;
} PerDeviceData;

bool explicitlyInitialized = false;

// Initialize kernel values
double a = 2.5;

// Normally you would want multiple warps, but to emphasize concurrency with streams and multiple devices.
// We run the kernels on a single warp.
int threadsPerBlock = 32;
int threadBlocks = 1;

// Configurable number of kernels. (streams, when running concurrently.)
int const numKernels = 4;
int const numStreams = numKernels;
vector<size_t> elements(numKernels);

// Each kernel call allocates and computes (call number) * (blockSize) elements.
// For 4 calls, this is 4k elements * 2 arrays * (1 + 2 + 3 + 4 stream mul) * 8B/elem =~ 640KB
int const blockSize = 4 * 1024;

// Macros
#define DAXPY_REPEAT 32768

// Kernels
// Loop over array of elements performing daxpy multiple times.
// To be launched with only one block (artificially increasing serial time to better demonstrate overlapping replay).
__global__ void
DaxpyKernel(
    int elements,
    double a,
    double *x,
    double *y)
{
    for (int i = threadIdx.x; i < elements; i += blockDim.x)
        // Artificially increase kernel runtime to emphasize concurrency.
        for (int j = 0; j < DAXPY_REPEAT; j++)
            y[i] = a * x[i] + y[i]; // daxpy
}

// Call any needed initialization routines for host or target.
void
ExplicitInitialization()
{
    if (explicitlyInitialized == false)
    {
        // CUPTI Profiler API initialization.
        CUpti_Profiler_Initialize_Params profilerInitializeParams = { CUpti_Profiler_Initialize_Params_STRUCT_SIZE };
        CUPTI_API_CALL(cuptiProfilerInitialize(&profilerInitializeParams));

        // NVPW required initialization.
        NVPW_InitializeHost_Params initializeHostParams = { NVPW_InitializeHost_Params_STRUCT_SIZE };
        NVPW_API_CALL(NVPW_InitializeHost(&initializeHostParams));

        explicitlyInitialized = true;
    }
}

/**
 * \brief Initialize Config, Counter Data Prefix, Counter Data, and Counter Data Scratch Buffer images.
 *
 * This should be run on the target.  In particular, counterAvailabilityImage, counterDataImage, and counterDataScratchBuffer
 * must be generated on the target.  ConfigImage and counterDataPrefixImage may be run on the host, but it may be simpler to do
 * all initialization in one place.
 *
 * @param config [in|out] config struct.  All fields except pChipName should be provided.  Returns same struct with pChipName filled.
 * @param MetricNames [in] vector of Perfworks API metric name strings.
 * @param configImage [out] Returns initialized Config image.
 * @param counterDataPrefixImage [out] Returns initialized Counter Data Prefix image.
 * @param counterDataImage [out] Returns initialized Counter Data image.
 * @param counterDataScratchBufferImage [out] Returns initiailized Counter Data Scratch Buffer image.
 */
void
TargetInitProfiling(
    PerDeviceData &deviceData,
    vector<string> const &MetricNames)
{
    // Ensure CUPTI Profiling API & NVPW are initialized.
    ExplicitInitialization();

    // Get size of counterAvailabilityImage - in first pass, GetCounterAvailability return size needed for data.
    CUpti_Profiler_GetCounterAvailability_Params getCounterAvailabilityParams = { CUpti_Profiler_GetCounterAvailability_Params_STRUCT_SIZE };
    getCounterAvailabilityParams.ctx = deviceData.config.context;
    CUPTI_API_CALL(cuptiProfilerGetCounterAvailability(&getCounterAvailabilityParams));

    // Allocate sized counterAvailabilityImage.
    vector<uint8_t> counterAvailabilityImage;
    counterAvailabilityImage.resize(getCounterAvailabilityParams.counterAvailabilityImageSize);

    // Initialize counterAvailabilityImage.
    getCounterAvailabilityParams.pCounterAvailabilityImage = counterAvailabilityImage.data();
    CUPTI_API_CALL(cuptiProfilerGetCounterAvailability(&getCounterAvailabilityParams));

    // Get chip name for the CUDA device.
    CUpti_Device_GetChipName_Params getChipNameParams = { CUpti_Device_GetChipName_Params_STRUCT_SIZE };
    getChipNameParams.deviceIndex = deviceData.config.device;
    CUPTI_API_CALL(cuptiDeviceGetChipName(&getChipNameParams));
    deviceData.config.pChipName = strdup(getChipNameParams.pChipName);

    // Fill in configImage - can be run on host or target.
    if (!NV::Metric::Config::GetConfigImage(deviceData.config.pChipName, MetricNames, deviceData.configImage, counterAvailabilityImage.data()))
    {
        cerr << "Failed to create configImage" << endl;
        exit(EXIT_FAILURE);
    }

    // Fill in counterDataPrefixImage - can be run on host or target.
    if (!NV::Metric::Config::GetCounterDataPrefixImage(deviceData.config.pChipName, MetricNames, deviceData.counterDataPrefixImage, counterAvailabilityImage.data()))
    {
        cerr << "Failed to create counterDataPrefixImage" << endl;
        exit(EXIT_FAILURE);
    }

    // Record counterDataPrefixImage info and other options for sizing the counterDataImage.
    CUpti_Profiler_CounterDataImageOptions counterDataImageOptions;
    counterDataImageOptions.pCounterDataPrefix = deviceData.counterDataPrefixImage.data();
    counterDataImageOptions.counterDataPrefixSize = deviceData.counterDataPrefixImage.size();
    counterDataImageOptions.maxNumRanges = deviceData.config.maxNumRanges;
    counterDataImageOptions.maxNumRangeTreeNodes = deviceData.config.maxNumRanges;
    counterDataImageOptions.maxRangeNameLength = deviceData.config.maxRangeNameLength;

    // Calculate size of counterDataImage based on counterDataPrefixImage and options.
    CUpti_Profiler_CounterDataImage_CalculateSize_Params calculateSizeParams = { CUpti_Profiler_CounterDataImage_CalculateSize_Params_STRUCT_SIZE };
    calculateSizeParams.pOptions = &counterDataImageOptions;
    calculateSizeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateSize(&calculateSizeParams));
    // Create counterDataImage
    deviceData.counterDataImage.resize(calculateSizeParams.counterDataImageSize);

    // Initialize counterDataImage.
    CUpti_Profiler_CounterDataImage_Initialize_Params initializeParams = { CUpti_Profiler_CounterDataImage_Initialize_Params_STRUCT_SIZE };
    initializeParams.pOptions = &counterDataImageOptions;
    initializeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    initializeParams.counterDataImageSize = deviceData.counterDataImage.size();
    initializeParams.pCounterDataImage = deviceData.counterDataImage.data();
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitialize(&initializeParams));

    // Calculate scratchBuffer size based on counterDataImage size and counterDataImage.
    CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params scratchBufferSizeParams = { CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params_STRUCT_SIZE };
    scratchBufferSizeParams.counterDataImageSize = deviceData.counterDataImage.size();
    scratchBufferSizeParams.pCounterDataImage = deviceData.counterDataImage.data();
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateScratchBufferSize(&scratchBufferSizeParams));
    // Create counterDataScratchBuffer.
    deviceData.counterDataScratchBufferImage.resize(scratchBufferSizeParams.counterDataScratchBufferSize);

    // Initialize counterDataScratchBuffer.
    CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params initScratchBufferParams = { CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params_STRUCT_SIZE };
    initScratchBufferParams.counterDataImageSize = deviceData.counterDataImage.size();
    initScratchBufferParams.pCounterDataImage = deviceData.counterDataImage.data();
    initScratchBufferParams.counterDataScratchBufferSize = deviceData.counterDataScratchBufferImage.size();;
    initScratchBufferParams.pCounterDataScratchBuffer = deviceData.counterDataScratchBufferImage.data();
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitializeScratchBuffer(&initScratchBufferParams));
}

void
StartSession(
    ProfilingConfig &config,
    vector<uint8_t> &counterDataImage,
    vector<uint8_t> &counterDataScratchBuffer,
    vector<uint8_t> &configImage)
{
    // Ensure CUPTI Profiling API & NVPW are initialized. (Only needed if not previously initialized.)
    ExplicitInitialization();

    // Start a session.
    CUpti_Profiler_BeginSession_Params beginSessionParams = { CUpti_Profiler_BeginSession_Params_STRUCT_SIZE };
    beginSessionParams.counterDataImageSize = counterDataImage.size();
    beginSessionParams.pCounterDataImage = counterDataImage.data();
    beginSessionParams.counterDataScratchBufferSize = counterDataScratchBuffer.size();
    beginSessionParams.pCounterDataScratchBuffer = counterDataScratchBuffer.data();
    beginSessionParams.ctx = config.context;
    beginSessionParams.maxLaunchesPerPass = config.maxLaunchesPerPass;
    beginSessionParams.maxRangesPerPass = config.maxRangesPerPass;
    beginSessionParams.pPriv = NULL;
    beginSessionParams.range = config.rangeMode;
    beginSessionParams.replayMode = config.replayMode;
    CUPTI_API_CALL(cuptiProfilerBeginSession(&beginSessionParams));

    CUpti_Profiler_SetConfig_Params setConfigParams = { CUpti_Profiler_SetConfig_Params_STRUCT_SIZE };
    setConfigParams.pConfig = configImage.data();
    setConfigParams.configSize = configImage.size();
    // Only set for Application Replay mode.
    setConfigParams.passIndex = 0;
    setConfigParams.minNestingLevel = config.minNestingLevels;
    setConfigParams.numNestingLevels = config.numNestingLevels;
    setConfigParams.targetNestingLevel = config.minNestingLevels;
    CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));
}

// Simple wrappers for Profiler API routines.
void
BeginPass(
    ProfilingConfig const &Config)
{
    CUpti_Profiler_BeginPass_Params beginPassParams = { CUpti_Profiler_BeginPass_Params_STRUCT_SIZE };
    beginPassParams.ctx = Config.context;
    CUPTI_API_CALL(cuptiProfilerBeginPass(&beginPassParams));
}

void
EnableProfiling(
    ProfilingConfig const &Config)
{
    CUpti_Profiler_EnableProfiling_Params enableProfilingParams = { CUpti_Profiler_EnableProfiling_Params_STRUCT_SIZE };
    enableProfilingParams.ctx = Config.context;
    CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
}

void PushRange(
    ProfilingConfig const &Config,
    char const *pRangeName)
{
    CUpti_Profiler_PushRange_Params pushRangeParams = { CUpti_Profiler_PushRange_Params_STRUCT_SIZE };
    pushRangeParams.ctx = Config.context;
    pushRangeParams.pRangeName = pRangeName;
    pushRangeParams.rangeNameLength = strlen(pRangeName);
    CUPTI_API_CALL(cuptiProfilerPushRange(&pushRangeParams));
}

void
PopRange(
    ProfilingConfig const &Config)
{
    CUpti_Profiler_PopRange_Params popRangeParams = { CUpti_Profiler_PopRange_Params_STRUCT_SIZE };
    popRangeParams.ctx = Config.context;
    CUPTI_API_CALL(cuptiProfilerPopRange(&popRangeParams));
}

void
DisableProfiling(
    ProfilingConfig &Config)
{
    CUpti_Profiler_DisableProfiling_Params disableProfilingParams = { CUpti_Profiler_DisableProfiling_Params_STRUCT_SIZE };
    disableProfilingParams.ctx = Config.context;
    CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));
}

bool
EndPass(
    ProfilingConfig &Config)
{
    CUpti_Profiler_EndPass_Params endPassParams = { CUpti_Profiler_EndPass_Params_STRUCT_SIZE };
    endPassParams.ctx = Config.context;
    CUPTI_API_CALL(cuptiProfilerEndPass(&endPassParams));
    return endPassParams.allPassesSubmitted;
}

void
EndSession(
    ProfilingConfig &Config)
{
    CUpti_Profiler_UnsetConfig_Params unsetConfigParams = { CUpti_Profiler_UnsetConfig_Params_STRUCT_SIZE };
    unsetConfigParams.ctx = Config.context;
    CUPTI_API_CALL(cuptiProfilerUnsetConfig(&unsetConfigParams));

    CUpti_Profiler_EndSession_Params endSessionParams = { CUpti_Profiler_EndSession_Params_STRUCT_SIZE };
    endSessionParams.ctx = Config.context;
    CUPTI_API_CALL(cuptiProfilerEndSession(&endSessionParams));
}

void
FlushData(
    ProfilingConfig &Config)
{
    CUpti_Profiler_FlushCounterData_Params flushCounterDataParams = { CUpti_Profiler_FlushCounterData_Params_STRUCT_SIZE };
    flushCounterDataParams.ctx = Config.context;

    CUPTI_API_CALL(cuptiProfilerFlushCounterData(&flushCounterDataParams));
    if (flushCounterDataParams.numRangesDropped != 0 || flushCounterDataParams.numTraceBytesDropped)
    {
        cerr << "WARNING: " << flushCounterDataParams.numTraceBytesDropped << " trace bytes dropped due to full TraceBuffer" << endl;
        cerr << "WARNING: " << flushCounterDataParams.numRangesDropped << " ranges dropped in pass" << endl;
    }
}

// Wrapper which will launch numKernel kernel calls on a single device.
// The device streams vector is used to control which stream each call is made on.
// If 'serial' is non-zero, the device streams are ignored and instead the default stream is used.
void
ProfileKernels(
    PerDeviceData &deviceData,
    char const * const RangeName,
    bool serial)
{
    // Switch to desired device
    RUNTIME_API_CALL(hipSetDevice(deviceData.deviceID));
    DRIVER_API_CALL(hipCtxSetCurrent(deviceData.config.context));

    // Use the same pass structure for multiple streams on a this device.
    StartSession(deviceData.config, deviceData.counterDataImage, deviceData.counterDataScratchBufferImage, deviceData.configImage);

    int numPasses = 0;
    bool lastPass = false;
    // Perform multiple passes if needed to provide all configured metrics.
    // Note that in this mode, kernel input data is not restored to initial values before each pass.
    do
    {
        BeginPass(deviceData.config);
        numPasses++;
        EnableProfiling(deviceData.config);

        // Then, time launching same amount of work in separate streams. (or default stream if serial.)
        // cuptiProfilerPushRange and PopRange will serialize the kernel launches, so keep the calls outside the concurrent stream launch loop.
        PushRange(deviceData.config, RangeName);

        for (unsigned int stream = 0; stream < deviceData.streams.size(); stream++)
        {
            hipStream_t streamId = (serial ? 0 : deviceData.streams[stream]);
            DaxpyKernel <<< threadBlocks, threadsPerBlock, 0, streamId >>> (elements[stream], a, deviceData.d_x[stream], deviceData.d_y[stream]);
            RUNTIME_API_CALL(hipGetLastError());

        }

        // After launching all work, synchronize all streams.
        if (serial == false)
        {
            for (unsigned int stream = 0; stream < deviceData.streams.size(); stream++)
            {
                RUNTIME_API_CALL(hipStreamSynchronize(deviceData.streams[stream]));
            }
        }
        else
        {
            RUNTIME_API_CALL(hipStreamSynchronize(0));
        }

        PopRange(deviceData.config);

        DisableProfiling(deviceData.config);

        lastPass = EndPass(deviceData.config);
    }
    while (lastPass == false);

    // Flush is required to ensure data is returned from device when running User Replay mode.
    FlushData(deviceData.config);

    EndSession(deviceData.config);
}


int
main(
    int argc,
    char *argv[])
{
    // These two metrics will demonstrate whether kernels within a Range were run serially or concurrently.
    vector<string> metricNames;
    metricNames.push_back("sm__cycles_active.sum");
    metricNames.push_back("sm__cycles_elapsed.max");
    // This metric shows that the same number of flops were executed on each run.
    metricNames.push_back("smsp__sass_thread_inst_executed_op_dfma_pred_on.sum");

    int numDevices;
    RUNTIME_API_CALL(hipGetDeviceCount(&numDevices));

    // Per-device information.
    vector<int> deviceIds;

    // Initialize profiler API support before testing device compatibility.
    ExplicitInitialization();

    // Find all devices capable of running CUPTI Profiling.
    for (int i = 0; i < numDevices; i++)
    {
        // Get device compatibility.
        CUpti_Profiler_DeviceSupported_Params params = { CUpti_Profiler_DeviceSupported_Params_STRUCT_SIZE };
        params.cuDevice = i;
        params.api = CUPTI_PROFILER_RANGE_PROFILING;
        CUPTI_API_CALL(cuptiProfilerDeviceSupported(&params));
        if (params.isSupported == CUPTI_PROFILER_CONFIGURATION_SUPPORTED)
        {
            // Record device number.
            deviceIds.push_back(i);
        }
        else
        {
            cerr << "Unable to profile on device " << i << ":" << endl;

            if (params.architecture == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
            {
                cerr << "\tDevice architecture is not supported" << endl;
            }

            if (params.sli == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
            {
                cerr << "\tDevice SLI configuration is not supported" << endl;
            }

            if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
            {
                cerr << "\tDevice VGPU configuration is not supported" << endl;
            }
            else if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_DISABLED)
            {
                cerr << "\tDevice VGPU configuration disabled profiling support" << endl;
            }

            if (params.confidentialCompute == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
            {
                cerr << "\tDevice Confidential Compute configuration is not supported" << endl;
            }

            if (params.cmp == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
            {
                cerr << "\tNVIDIA Crypto Mining Processors (CMP) are not supported" << endl;
            }

            if (params.wsl == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED)
            {
                cerr << "\tWSL is not supported" << endl;
            }
        }
    }

    numDevices = deviceIds.size();
    cout << "Found " << numDevices << " compatible devices" << endl;

    // Ensure we found at least one device.
    if (numDevices == 0)
    {
        cerr << "No devices detected compatible with CUPTI Profiling" << endl;
        exit(EXIT_WAIVED);
    }

    // Initialize kernel input to some known numbers.
    vector<double> h_x(blockSize * numKernels);
    vector<double> h_y(blockSize * numKernels);
    for (size_t i = 0; i < blockSize * numKernels; i++)
    {
        h_x[i] = 1.5 * i;
        h_y[i] = 2.0 * (i - 3000);
    }

    // Initialize a vector of 'default stream' values to demonstrate serialized kernels.
    vector<hipStream_t> defaultStreams(numStreams);
    for (int stream = 0; stream < numStreams; stream++)
    {
        defaultStreams[stream] = 0;
    }

    // Scale per-kernel work by stream number.
    for (int stream = 0; stream < numStreams; stream++)
    {
        elements[stream] = blockSize * (stream + 1);
    }

    // For each device, configure profiling, set up buffers, copy kernel data.
    vector<PerDeviceData> deviceData(numDevices);

    for (int device = 0; device < numDevices; device++)
    {
        int deviceId = deviceIds[device];
        RUNTIME_API_CALL(hipSetDevice(deviceId));
        cout << "Configuring device " << deviceId << endl;

        // Required CUPTI Profiling configuration & initialization.
        // Can be done ahead of time or immediately before startSession() call.
        // Initialization & configuration images can be generated separately, then passed to later calls.
        // For simplicity's sake, in this sample, a single config struct is created per device and passed to each CUPTI Profiler API call.
        // For more complex cases, each combination of CUPTI Profiler Session and Config requires additional initialization.
        ProfilingConfig config;
        // Device ID, used to get device name for metrics enumeration.
        config.device = deviceId;
        // Must be >= maxRangesPerPass.  Set this to the largest count of kernel launches which may be encountered in any Pass in this Session.
        config.maxLaunchesPerPass = 1;

        // Device 0 has max of 3 passes; other devices only run one pass in this sample code
        if (device == 0)
        {
            // Maximum number of ranges that may be profiled in the current Session
            config.maxNumRanges = 3;
        }
        else
        {
            // Maximum number of ranges that may be profiled in the current Session
            config.maxNumRanges = 1;
        }

        // Max length including NULL terminator of any range name.
        config.maxRangeNameLength = 64;
        // Max ranges that can be recorded in any Pass in this Session.
        config.maxRangesPerPass = 1;
        // Must be >= 1, minimum reported nest level for Ranges in this Session.
        config.minNestingLevels = 1;
        // Must be >= 1, max height of nested Ranges in this Session.
        config.numNestingLevels = 1;
        // CUPTI_AutoRange or CUPTI_UserRange.
        config.rangeMode = CUPTI_UserRange;
        // CUPTI_KernelReplay, CUPTI_UserReplay, or CUPTI_ApplicationReplay.
        config.replayMode = CUPTI_UserReplay;
        // Either set to a context, or may be NULL if a default context has been created.
        DRIVER_API_CALL(hipCtxCreate(&(config.context), 0, device));
        // Save this device config.
        deviceData[device].config = config;

        // Initialize CUPTI Profiling structures.
        TargetInitProfiling(deviceData[device], metricNames);

        // Per-stream initialization & memory allocation - copy from constant host array to each device array.
        deviceData[device].streams.resize(numStreams);
        deviceData[device].d_x.resize(numStreams);
        deviceData[device].d_y.resize(numStreams);
        for (int stream = 0; stream < numStreams; stream++)
        {
            RUNTIME_API_CALL(hipStreamCreate(&(deviceData[device].streams[stream])));

            // Each kernel does (stream #) * blockSize work on doubles.
            size_t size = elements[stream] * sizeof(double);

            RUNTIME_API_CALL(hipMalloc(&(deviceData[device].d_x[stream]), size));
            MEMORY_ALLOCATION_CALL(deviceData[device].d_x[stream]);
            RUNTIME_API_CALL(hipMemcpy(deviceData[device].d_x[stream], h_x.data(), size, hipMemcpyHostToDevice));

            RUNTIME_API_CALL(hipMalloc(&(deviceData[device].d_y[stream]), size));
            MEMORY_ALLOCATION_CALL(deviceData[device].d_y[stream]);
            RUNTIME_API_CALL(hipMemcpy(deviceData[device].d_y[stream], h_x.data(), size, hipMemcpyHostToDevice));
        }
    }

    // First version - single device, kernel calls serialized on default stream.
    // Use wallclock time to measure performance.
    auto begin_time = ::std::chrono::high_resolution_clock::now();

    // Run on first device and use default streams, which run serially.
    ProfileKernels(deviceData[0], "single_device_serial", true);

    auto end_time = ::std::chrono::high_resolution_clock::now();
    auto elapsed_serial_ms = ::std::chrono::duration_cast<::std::chrono::milliseconds>(end_time - begin_time);
    int numBlocks = 0;
    for (int i = 1; i <= numKernels; i++)
    {
        numBlocks += i;
    }
    cout << "It took " << elapsed_serial_ms.count() << "ms on the host to profile " << numKernels << " kernels in serial." << endl;

    // Second version - same kernel calls as before on the same device, but now using separate streams for concurrency.
    // (Should be limited by the longest running kernel.)

    begin_time = ::std::chrono::high_resolution_clock::now();

    // Still only use first device, but this time use its allocated streams for parallelism.
    ProfileKernels(deviceData[0], "single_device_async", false);

    end_time = ::std::chrono::high_resolution_clock::now();
    auto elapsed_single_device_ms = ::std::chrono::duration_cast<::std::chrono::milliseconds>(end_time - begin_time);
    cout << "It took " << elapsed_single_device_ms.count() << "ms on the host to profile " << numKernels << " kernels on a single device on separate streams." << endl;
    cout << "--> If the separate stream wallclock time is less than the serial version, the streams were profiling concurrently." << endl;

    // Third version - same as the second case, but duplicates the concurrent work across devices to show cross-device concurrency.
    // This is done using devices so no serialization is needed between devices.
    // (Should have roughly the same wallclock time as second case if the devices have similar performance)

    if (numDevices == 1)
    {
        cout << "Only one compatible device found; skipping the multi-threaded test." << endl;
    }
    else
    {
        cout << "Running on " << numDevices << " devices, one thread per device." << endl;

        // Time creation of the same multiple streams. (on multiple devices, if possible.)
        vector<::std::thread> threads;
        begin_time = ::std::chrono::high_resolution_clock::now();

        // Now launch parallel thread work, duplicated on one thread per device.
        for (int thread = 0; thread < numDevices; thread++)
        {
            threads.push_back(::std::thread(ProfileKernels, ::std::ref(deviceData[thread]), "multi_device_async", false));
        }

        // Wait for all threads to finish.
        for (auto &t: threads)
        {
            t.join();
        }

        // Record time used when launching on multiple devices.
        end_time = ::std::chrono::high_resolution_clock::now();
        auto elapsed_multiple_device_ms = ::std::chrono::duration_cast<::std::chrono::milliseconds>(end_time - begin_time);
        cout << "It took " << elapsed_multiple_device_ms.count() << "ms on the host to profile the same " << numKernels << " kernels on each of the " << numDevices << " devices in parallel" << endl;
        cout << "--> Wallclock ratio of parallel device launch to single device launch is " << elapsed_multiple_device_ms.count() / static_cast<double>(elapsed_single_device_ms.count()) << endl;
        cout << "--> If the ratio is close to 1, that means there was little overhead to profile in parallel on multiple devices compared to profiling on a single device." << endl;
        cout << "--> If the devices have different performance, the ratio may not be close to one, and this should be limited by the slowest device." << endl;
    }

    // Free stream memory for each device.
    for (int i = 0; i < numDevices; i++)
    {
        for (int j = 0; j < numKernels; j++)
        {
            RUNTIME_API_CALL(hipFree(deviceData[i].d_x[j]));
            RUNTIME_API_CALL(hipFree(deviceData[i].d_y[j]));
        }
    }

    // Display metric values.
    cout << endl << "Metrics for device #0:" << endl;
    cout << "Look at the sm__cycles_elapsed.max values for each test." << endl;
    cout << "This value represents the time spent on device to run the kernels in each case, and should be longest for the serial range, and roughly equal for the single and multi device concurrent ranges." << endl;
    PrintMetricValues(deviceData[0].config.pChipName, deviceData[0].counterDataImage, metricNames);

    // Only display next device info if needed.
    if (numDevices > 1)
    {
        cout << endl << "Metrics for the remaining devices only display the multi device async case and should all be similar to the first device's values if the device has similar performance characteristics." << endl;
        cout << "If devices have different performance characteristics, the runtime cycles calculation may vary by device." << endl;
    }
    for (int i = 1; i < numDevices; i++)
    {
        cout << endl << "Metrics for device #" << i << ":" << endl;
        PrintMetricValues(deviceData[i].config.pChipName, deviceData[i].counterDataImage, metricNames);
    }

    exit(EXIT_SUCCESS);
}
